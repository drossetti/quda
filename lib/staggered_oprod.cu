#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <staggered_oprod.h>

#include <tune_quda.h>
#include <quda_internal.h>
#include <gauge_field_order.h>
#include <quda_matrix.h>

namespace quda {

  namespace { // anonymous
#include <texture.h>
  }

  static bool kernelPackT = true;

  template<int N>
    void createEventArray(hipEvent_t (&event)[N], unsigned int flags=hipEventDefault)
    {
      for(int i=0; i<N; ++i)
        hipEventCreate(&event[i],flags);
      return;
    }

  template<int N>
    void destroyEventArray(hipEvent_t (&event)[N])
    {
      for(int i=0; i<N; ++i)
        hipEventDestroy(event[i]);
    }


  static hipEvent_t packEnd;
  static hipEvent_t gatherEnd[4];
  static hipEvent_t scatterEnd[4];
  static hipEvent_t oprodStart;
  static hipEvent_t oprodEnd;


  void createStaggeredOprodEvents(){
#ifdef MULTI_GPU
    hipEventCreate(&packEnd, hipEventDisableTiming);
    createEventArray(gatherEnd, hipEventDisableTiming);
    createEventArray(scatterEnd, hipEventDisableTiming);
#endif
    hipEventCreate(&oprodStart, hipEventDisableTiming);
    hipEventCreate(&oprodEnd, hipEventDisableTiming);
    return;
  }

  void destroyStaggeredOprodEvents(){
#ifdef MULTI_GPU
    destroyEventArray(gatherEnd);
    destroyEventArray(scatterEnd);
    hipEventDestroy(packEnd);
#endif
    hipEventDestroy(oprodStart);
    hipEventDestroy(oprodEnd);
    return;
  }


  enum KernelType {OPROD_INTERIOR_KERNEL, OPROD_EXTERIOR_KERNEL};

  template<typename Complex, typename Output, typename Input>
    struct StaggeredOprodArg {
      unsigned int length;
      unsigned int X[4];
      unsigned int parity;
      unsigned int dir;
      unsigned int ghostOffset;
      unsigned int displacement;
      KernelType kernelType;
      bool partitioned[4];
      Input inA;
      Input inB;
      Output outA;
      Output outB;
      typename RealTypeId<Complex>::Type coeff[2];

      StaggeredOprodArg(const unsigned int length,
          const unsigned int X[4],
          const unsigned int parity,
          const unsigned int dir,
          const unsigned int ghostOffset,
          const unsigned int displacement,   
          const KernelType& kernelType, 
          const double coeff[2],
          Input& inA,
          Input& inB,
          Output& outA,
          Output& outB) : length(length), parity(parity), ghostOffset(ghostOffset), 
      displacement(displacement), kernelType(kernelType), inA(inA), inB(inB), outA(outA), outB(outB) 
      {
        this->coeff[0] = coeff[0];
        this->coeff[1] = coeff[1];
        for(int i=0; i<4; ++i) this->X[i] = X[i];
        for(int i=0; i<4; ++i) this->partitioned[i] = commDimPartitioned(i) ? true : false;
      }
    };

  enum IndexType {
    EVEN_X = 0,
    EVEN_Y = 1,
    EVEN_Z = 2,
    EVEN_T = 3
  };

  template <IndexType idxType>
    static __device__ __forceinline__ void coordsFromIndex(int& idx, int c[4],  
        const unsigned int cb_idx, const unsigned int parity, const unsigned int X[4])
    {
      const unsigned int &LX = X[0];
      const unsigned int &LY = X[1];
      const unsigned int &LZ = X[2];
      const unsigned int XYZ = X[2]*X[1]*X[0];
      const unsigned int XY = X[1]*X[0];

      idx = 2*cb_idx;

      int x, y, z, t;

      if (idxType == EVEN_X /*!(LX & 1)*/) { // X even
        //   t = idx / XYZ;
        //   z = (idx / XY) % Z;
        //   y = (idx / X) % Y;
        //   idx += (parity + t + z + y) & 1;
        //   x = idx % X;
        // equivalent to the above, but with fewer divisions/mods:
        int aux1 = idx / LX;
        x = idx - aux1 * LX;
        int aux2 = aux1 / LY;
        y = aux1 - aux2 * LY;
        t = aux2 / LZ;
        z = aux2 - t * LZ;
        aux1 = (parity + t + z + y) & 1;
        x += aux1;
        idx += aux1;
      } else if (idxType == EVEN_Y /*!(LY & 1)*/) { // Y even
        t = idx / XYZ;
        z = (idx / XY) % LZ;
        idx += (parity + t + z) & 1;
        y = (idx / LX) % LY;
        x = idx % LX;
      } else if (idxType == EVEN_Z /*!(LZ & 1)*/) { // Z even
        t = idx / XYZ;
        idx += (parity + t) & 1;
        z = (idx / XY) % LZ;
        y = (idx / LX) % LY;
        x = idx % LX;
      } else {
        idx += parity;
        t = idx / XYZ;
        z = (idx / XY) % LZ;
        y = (idx / LX) % LY;
        x = idx % LX;
      }

      c[0] = x;
      c[1] = y;
      c[2] = z;
      c[3] = t;
    }




  // Get the  coordinates for the exterior kernels
  template<int Nspin>
    __device__ void coordsFromIndex(unsigned int x[4], const unsigned int cb_idx, const unsigned int X[4], const unsigned int dir, const int displacement, const unsigned int parity)
    {

      if(Nspin == 1){
        unsigned int Xh[2] = {X[0]/2, X[1]/2};
        switch(dir){
          case 0:
            x[2] = cb_idx/Xh[1] % X[2];
            x[3] = cb_idx/(Xh[1]*X[2]) % X[3];
            x[0] = cb_idx/(Xh[1]*X[2]*X[3]);
            x[0] += (X[0] - displacement);
            x[1] = 2*(cb_idx % Xh[1]) + ((x[0]+x[2]+x[3]+parity)&1);
            break;

          case 1:
            x[2] = cb_idx/Xh[0] % X[2];
            x[3] = cb_idx/(Xh[0]*X[2]) % X[3];
            x[1] = cb_idx/(Xh[0]*X[2]*X[3]);
            x[1] += (X[1] - displacement);
            x[0] = 2*(cb_idx % Xh[0]) + ((x[1]+x[2]+x[3]+parity)&1);

            break;

          case 2:
            x[1] = cb_idx/Xh[0] % X[1];
            x[3] = cb_idx/(Xh[0]*X[1]) % X[3];
            x[2] = cb_idx/(Xh[0]*X[1]*X[3]);
            x[2] += (X[2] - displacement);
            x[0] = 2*(cb_idx % Xh[0]) + ((x[1]+x[2]+x[3]+parity)&1);

            break;

          case 3:
            x[1] = cb_idx/Xh[0] % X[1];
            x[2] = cb_idx/(Xh[0]*X[1]) % X[2];
            x[3] = cb_idx/(Xh[0]*X[1]*X[2]);
            x[3] += (X[3] - displacement);
            x[0] = 2*(cb_idx % Xh[0]) + ((x[1]+x[2]+x[3]+parity)&1);

            break;
        }
      }else if(Nspin == 3){
        // currently unsupported
      }
      return;
    }


  template<int Nspin, int Nface> 
    __device__  int ghostIndexFromCoords(const unsigned int x[4], const unsigned int X[4], const unsigned int dir, const int shift){
      return 0;
    }



  template<>
    __device__  int ghostIndexFromCoords<3,3>(
        const unsigned int x[4],
        const unsigned int X[4], 
        unsigned int dir, 
        const int shift)
    {
      unsigned int ghost_idx;
      if(shift > 0){
        if((x[dir] + shift) >= X[dir]){
          switch(dir){
            case 0:
              ghost_idx = (3*3 + (x[0]-X[0]+shift))*(X[3]*X[2]*X[1])/2 + ((x[3]*X[2] + x[2])*X[1] + x[1])/2;
              break;          
            case 1:
              ghost_idx = (3*3 + (x[1]-X[1]+shift))*(X[3]*X[2]*X[0])/2 + (x[3]*X[2]*X[0] + x[2]*X[0] + x[0])/2;
              break;
            case 2:
              ghost_idx = (3*3 + (x[2]-X[2]+shift))*(X[3]*X[1]*X[0])/2 + (x[3]*X[1]*X[0] + x[1]*X[0] + x[0])/2;
              break;
            case 3:
              ghost_idx = (3*3 + (x[3]-X[3]+shift))*(X[2]*X[1]*X[0])/2 + (x[2]*X[1]*X[0] + x[1]*X[0] + x[0])/2;
              break;
            default:
              break;
          } // switch
        } // x[dir] + shift[dir] >= X[dir]
      }else{ // shift < 0
        if(static_cast<int>(x[dir]) + shift < 0){
          switch(dir){
            case 0:
              ghost_idx = (3 + shift)*(X[3]*X[2]*X[1])/2 + ((x[3]*X[2] + x[2])*X[1] + x[1])/2;
              break;
            case 1:
              ghost_idx = (3 + shift)*(X[3]*X[2]*X[0])/2 + ((x[3]*X[2] + x[2])*X[0] + x[0])/2;
              break;
            case 2:
              ghost_idx = (3 + shift)*(X[3]*X[1]*X[0])/2 + ((x[3]*X[1] + x[1])*X[0]  + x[0])/2;
              break;
            case 3:
              ghost_idx = (3 + shift)*(X[2]*X[1]*X[0])/2 + ((x[2]*X[1] + x[1])*X[0] + x[0])/2;
              break;
          } // switch(dir)
        }
      } // shift < 0

      return ghost_idx;
    }




  __device__ __forceinline__
    int neighborIndex(const unsigned int& cb_idx, const int shift[4],  const bool partitioned[4], const unsigned int& parity, 
        const unsigned int X[4]){

      int  full_idx;
      int x[4]; 


      coordsFromIndex<EVEN_X>(full_idx, x, cb_idx, parity, X);

#ifdef MULTI_GPU
      for(int dim = 0; dim<4; ++dim){
        if(partitioned[dim])
          if( (x[dim]+shift[dim])<0 || (x[dim]+shift[dim])>=X[dim]) return -1;
      }
#endif

      for(int dim=0; dim<4; ++dim){
        x[dim] = shift[dim] ? (x[dim]+shift[dim] + X[dim]) % X[dim] : x[dim];
      }
      return  (((x[3]*X[2] + x[2])*X[1] + x[1])*X[0] + x[0]) >> 1;
    }



  template<typename Complex, typename Output, typename Input>
    __global__ void interiorOprodKernel(StaggeredOprodArg<Complex, Output, Input> arg)
    {
      unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
      const unsigned int gridSize = gridDim.x*blockDim.x;

      typedef typename RealTypeId<Complex>::Type real;
      Complex x[3];
      Complex y[3];
      Complex z[3];
      Matrix<Complex,3> result;
      Matrix<Complex,3> tempA, tempB; // input


      while(idx<arg.length){
        arg.inA.load(x, idx);
        for(int dir=0; dir<4; ++dir){
          int shift[4] = {0,0,0,0};
          shift[dir] = 1;
          const int first_nbr_idx = neighborIndex(idx, shift, arg.partitioned, arg.parity, arg.X);
          if(first_nbr_idx >= 0){
            arg.inB.load(y, first_nbr_idx);
            outerProd(y,x,&result);
            arg.outA.load(reinterpret_cast<real*>(tempA.data), idx, dir, arg.parity); 
            result = tempA + result*arg.coeff[0];
            arg.outA.save(reinterpret_cast<real*>(result.data), idx, dir, arg.parity); 

            shift[dir] = 3;
            const int third_nbr_idx = neighborIndex(idx, shift, arg.partitioned, arg.parity, arg.X);
            if(third_nbr_idx >= 0){
              arg.inB.load(z, third_nbr_idx);
              outerProd(z, x, &result);
              arg.outB.load(reinterpret_cast<real*>(tempB.data), idx, dir, arg.parity); 
              result = tempB + result*arg.coeff[1];
              arg.outB.save(reinterpret_cast<real*>(result.data), idx, dir, arg.parity); 
            }
          }
        } // dir
        idx += gridSize;
      }
      return;
    } // interiorOprodKernel



  template<typename Complex, typename Output, typename Input> 
    __global__ void exteriorOprodKernel(StaggeredOprodArg<Complex, Output, Input> arg)
    {
      unsigned int cb_idx = blockIdx.x*blockDim.x + threadIdx.x;
      const unsigned int gridSize = gridDim.x*blockDim.x;

      Complex a[3];
      Complex b[3];
      Matrix<Complex,3> result;
      Matrix<Complex,3> inmatrix; // input
      typedef typename RealTypeId<Complex>::Type real;


      Output& out = (arg.displacement == 1) ? arg.outA : arg.outB;
      real coeff = (arg.displacement == 1) ? arg.coeff[0] : arg.coeff[1];

      unsigned int x[4];
      while(cb_idx<arg.length){
        coordsFromIndex<1>(x, cb_idx, arg.X, arg.dir, arg.displacement, arg.parity); 
        const unsigned int bulk_cb_idx = ((((x[3]*arg.X[2] + x[2])*arg.X[1] + x[1])*arg.X[0] + x[0]) >> 1);

        out.load(reinterpret_cast<real*>(inmatrix.data), bulk_cb_idx, arg.dir, arg.parity); 
        arg.inA.load(a, bulk_cb_idx);

        const unsigned int ghost_idx = arg.ghostOffset + ghostIndexFromCoords<3,3>(x, arg.X, arg.dir, arg.displacement);
        arg.inB.load(b, ghost_idx);

        outerProd(b,a,&result);
        result = inmatrix + result*coeff; 
        out.save(reinterpret_cast<real*>(result.data), bulk_cb_idx, arg.dir, arg.parity); 

        cb_idx += gridSize;
      }
      return;
    }



  template<typename Complex, typename Output, typename Input> 
    class StaggeredOprodField : public Tunable {

      private:
        StaggeredOprodArg<Complex,Output,Input> arg;
        QudaFieldLocation location; // location of the lattice fields

        unsigned int sharedBytesPerThread() const { return 0; }
        unsigned int sharedBytesPerBlock(const TuneParam &) const { return 0; }

        unsigned int minThreads() const { return arg.outA.volumeCB; }
        bool tunedGridDim() const { return false; }

      public:
        StaggeredOprodField(const StaggeredOprodArg<Complex,Output,Input> &arg,
            QudaFieldLocation location)
          : arg(arg), location(location) {} 

        virtual ~StaggeredOprodField() {}

        void set(const StaggeredOprodArg<Complex,Output,Input> &arg, QudaFieldLocation location){
          // This is a hack. Need to change this!
          this->arg.dir = arg.dir;
          this->arg.length = arg.length;
          this->arg.ghostOffset = arg.ghostOffset;
          this->arg.kernelType = arg.kernelType;
          this->location = location;
        } // set

        void apply(const hipStream_t &stream){
          if(location == QUDA_CUDA_FIELD_LOCATION){
            // Disable tuning for the time being
            TuneParam tp;
            // TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
            if(arg.kernelType == OPROD_INTERIOR_KERNEL){
              //interiorOprodKernel<<<tp.grid,tp.block,tp.shared_bytes, stream>>>(arg);
              dim3 blockDim(128, 1, 1);
              const int gridSize = (arg.length + (blockDim.x-1))/blockDim.x;
              dim3 gridDim(gridSize, 1, 1);               
              interiorOprodKernel<<<gridDim,blockDim,0, stream>>>(arg);
            }else if(arg.kernelType == OPROD_EXTERIOR_KERNEL){
              const unsigned int volume = arg.X[0]*arg.X[1]*arg.X[2]*arg.X[3];
              arg.inB.setStride(3*volume/(2*arg.X[arg.dir]));
              exteriorOprodKernel<<<tp.grid,tp.block,tp.shared_bytes, stream>>>(arg);
              arg.inB.setStride(arg.inA.Stride());
            }else{
              errorQuda("Kernel type not supported\n");
            }
          }else{ // run the CPU code
            errorQuda("No CPU support for staggered outer-product calculation\n");
          }
        } // apply

        void preTune(){}
        void postTune(){}

        long long flops() const {
          return 0; // fix this
        }

        long long bytes() const { 
          return 0; // fix this
        }

        TuneKey tuneKey() const {
          std::stringstream vol, aux;
          vol << arg.X[0] << "x";
          vol << arg.X[1] << "x";
          vol << arg.X[2] << "x";
          vol << arg.X[3] << "x";

          aux << "threads=" << arg.length << ",prec=" << sizeof(Complex)/2;
          aux << "stride=" << arg.inA.Stride();
          return TuneKey(vol.str(), typeid(*this).name(), aux.str());
        }
    }; // StaggeredOprodField

  template<typename Complex, typename Output, typename Input>
    void computeStaggeredOprodCuda(Output outA, Output outB, Input& inA, Input& inB, cudaColorSpinorField& src, 
        FaceBuffer& faceBuffer,  const unsigned int parity, const int faceVolumeCB[4], 
        const unsigned int ghostOffset[4], const double coeff[2])
    {

      hipEventRecord(oprodStart, streams[Nstream-1]);


      const unsigned int dim[4] = {src.X(0)*2, src.X(1), src.X(2), src.X(3)};
      // Create the arguments for the interior kernel 
      StaggeredOprodArg<Complex,Output,Input> arg(outA.volumeCB, dim, parity, 0, 0, 1, OPROD_INTERIOR_KERNEL, coeff, inA, inB, outA, outB);


      StaggeredOprodField<Complex,Output,Input> oprod(arg, QUDA_CUDA_FIELD_LOCATION);

#ifdef MULTI_GPU
      bool pack=false;
      for(int i=3; i>=0; i--){
        if(commDimPartitioned(i) && (i!=3 || kernelPackT)){
          pack = true;
          break;
        }
      } // i=3,..,0

      // source, dir(+/-1), parity, dagger, stream_ptr
      if(pack){
        faceBuffer.pack(src, -1, 1-parity, 0, streams); // packing is all done in streams[Nstream-1]
        //faceBuffer.pack(src, 1-parity, 0, streams); // packing is all done in streams[Nstream-1]
        hipEventRecord(packEnd, streams[Nstream-1]);
      }

      for(int i=3; i>=0; i--){
        if(commDimPartitioned(i)){

          hipEvent_t &event = (i!=3 || kernelPackT) ? packEnd : oprodStart;
          hipStreamWaitEvent(streams[2*i], event, 0); // wait in stream 2*i for event to complete
      

          // Initialize the host transfer from the source spinor
          faceBuffer.gather(src, false, 2*i); 
          // record the end of the gathering 
          hipEventRecord(gatherEnd[i], streams[2*i]);
        } // comDim(i)
      } // i=3,..,0
#endif
      oprod.apply(streams[Nstream-1]); 

#ifdef MULTI_GPU
      // compute gather completed 
      int gatherCompleted[5];
      int commsCompleted[5];
      int oprodCompleted[4];

      for(int i=0; i<4; ++i){
        gatherCompleted[i] = commsCompleted[i] = oprodCompleted[i] = 0;
      }
      gatherCompleted[4] = commsCompleted[4] = 1;

      // initialize commDimTotal 
      int commDimTotal = 0;
      for(int i=0; i<4; ++i){
        commDimTotal += commDimPartitioned(i);
      }
      commDimTotal *= 2;

      // initialize previousDir
      int previousDir[4];
      for(int i=3; i>=0; i--){
        if(commDimPartitioned(i)){
          int prev = 4;
          for(int j=3; j>i; j--){
            if(commDimPartitioned(j)){
              prev = j;
            }
          }
          previousDir[i] = prev;
        }
      } // set previous directions


      if(commDimTotal){
        arg.kernelType = OPROD_EXTERIOR_KERNEL;
        unsigned int completeSum=0;
        while(completeSum < commDimTotal){

          for(int i=3; i>=0; i--){
            if(!commDimPartitioned(i)) continue;

            if(!gatherCompleted[i] && gatherCompleted[previousDir[i]]){
              hipError_t event_test = hipEventQuery(gatherEnd[i]);

              if(event_test == hipSuccess){
                gatherCompleted[i] = 1;
                completeSum++;
                faceBuffer.commsStart(2*i);
              }
            }

            // Query if comms has finished 
            if(!commsCompleted[i] && commsCompleted[previousDir[i]] && gatherCompleted[i]){
              int comms_test = faceBuffer.commsQuery(2*i);
              if(comms_test){
                commsCompleted[i] = 1;
                completeSum++;
                faceBuffer.scatter(src, false, 2*i);
              }
            }

            // enqueue the boundary oprod kernel as soon as the scatters have been enqueud
            if(!oprodCompleted[i] && commsCompleted[i]){
              hipEventRecord(scatterEnd[i], streams[2*i]);
              hipStreamWaitEvent(streams[Nstream-1], scatterEnd[i],0);

              arg.dir = i;
              arg.ghostOffset = ghostOffset[i];
              const unsigned int volume = arg.X[0]*arg.X[1]*arg.X[2]*arg.X[3];
              arg.inB.setStride(3*volume/(2*arg.X[arg.dir]));
              // First, do the one hop term
              {

                arg.length = faceVolumeCB[i];
                arg.displacement = 1;
                dim3 blockDim(128, 1, 1);
                const int gridSize = (arg.length + (blockDim.x-1))/blockDim.x;
                dim3 gridDim(gridSize, 1, 1);               
                exteriorOprodKernel<<<gridDim, blockDim, 0, streams[Nstream-1]>>>(arg);              
              }
              // Now do the 3 hop term - Try putting this in a separate stream
              {

                arg.displacement = 3;                      
                arg.length = arg.displacement*faceVolumeCB[i];
                dim3 blockDim(128, 1, 1);
                const int gridSize = (arg.length + (blockDim.x-1))/blockDim.x;
                dim3 gridDim(gridSize, 1, 1);               
                exteriorOprodKernel<<<gridDim, blockDim, 0, streams[Nstream-1]>>>(arg);              
              } 
              arg.inB.setStride(arg.inA.Stride());

              oprodCompleted[i] = 1;
            }

          } // i=3,..,0 
        } // completeSum < commDimTotal
      } // if commDimTotal
#endif
    } // computeStaggeredOprodCuda


  // At the moment, I pass an instance of FaceBuffer in. 
  // Soon, faceBuffer will be subsumed into cudaColorSpinorField.
  void computeStaggeredOprod(cudaGaugeField& outA, cudaGaugeField& outB, cudaColorSpinorField& in,  
      FaceBuffer& faceBuffer,
      const unsigned int parity, const double coeff[2])
  {

    if(outA.Order() != QUDA_FLOAT2_GAUGE_ORDER)
      errorQuda("Unsupported output ordering: %d\n", outA.Order());    

    if(outB.Order() != QUDA_FLOAT2_GAUGE_ORDER)
      errorQuda("Unsupported output ordering: %d\n", outB.Order());    

    unsigned int ghostOffset[4] = {0,0,0,0};
#ifdef MULTI_GPU
    const unsigned int Npad = in.Ncolor()*in.Nspin()*2/in.FieldOrder();
    for(int dir=0; dir<4; ++dir){
      ghostOffset[dir] = Npad*(in.GhostOffset(dir) + in.Stride()); 
    }
#endif

    if(in.Precision() != outA.Precision()) errorQuda("Mixed precision not supported: %d %d\n", in.Precision(), outA.Precision());

    cudaColorSpinorField& inA = (parity&1) ? in.Odd() : in.Even();
    cudaColorSpinorField& inB = (parity&1) ? in.Even() : in.Odd();

    if(in.Precision() == QUDA_DOUBLE_PRECISION){

      Spinor<double2, double2, double2, 3, 0, 0> spinorA(inA);
      Spinor<double2, double2, double2, 3, 0, 0> spinorB(inB);
      computeStaggeredOprodCuda<double2>(FloatNOrder<double, 18, 2, 18>(outA), FloatNOrder<double, 18, 2, 18>(outB), 
          spinorA, spinorB, inB, faceBuffer, parity, inB.GhostFace(), ghostOffset, coeff);
    }else if(in.Precision() == QUDA_SINGLE_PRECISION){

      Spinor<float2, float2, float2, 3, 0, 0> spinorA(inA);
      Spinor<float2, float2, float2, 3, 0, 0> spinorB(inB);
      computeStaggeredOprodCuda<float2>(FloatNOrder<float, 18, 2, 18>(outA), FloatNOrder<float, 18, 2, 18>(outB), 
          spinorA, spinorB, inB, faceBuffer, parity, inB.GhostFace(), ghostOffset, coeff);
    }else{
      errorQuda("Unsupported precision: %d\n", in.Precision());
    }
    return;
  } // computeStaggeredOprod



} // namespace quda
