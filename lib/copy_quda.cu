#include "hip/hip_runtime.h"
#include <blas_quda.h>
#include <tune_quda.h>
#include <float_vector.h>

// For kernels with precision conversion built in
#define checkSpinorLength(a, b)						\
  {									\
    if (a.Length() != b.Length())					\
      errorQuda("lengths do not match: %d %d", a.Length(), b.Length());	\
    if (a.Stride() != b.Stride())					\
      errorQuda("strides do not match: %d %d", a.Stride(), b.Stride());	\
  }

namespace quda {

  hipStream_t* getBlasStream();
    
  namespace copy {

#include <texture.h>

    static struct {
      int x[QUDA_MAX_DIM];
      int stride;
    } blasConstants;

    template <typename FloatN, int N, typename Output, typename Input>
    __global__ void copyKernel(Output Y, Input X, int length) {
      unsigned int i = blockIdx.x*(blockDim.x) + threadIdx.x;
      unsigned int gridSize = gridDim.x*blockDim.x;

      while (i < length) {
	FloatN x[N];
	X.load(x, i);
	Y.save(x, i);
	i += gridSize;
      }
    }

    template <typename FloatN, int N, typename Output, typename Input>
    class CopyCuda : public Tunable {

    private:
      Input &X;
      Output &Y;
      const int length;

      unsigned int sharedBytesPerThread() const { return 0; }
      unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }

      virtual bool advanceSharedBytes(TuneParam &param) const
      {
	TuneParam next(param);
	advanceBlockDim(next); // to get next blockDim
	int nthreads = next.block.x * next.block.y * next.block.z;
	param.shared_bytes = sharedBytesPerThread()*nthreads > sharedBytesPerBlock(param) ?
	  sharedBytesPerThread()*nthreads : sharedBytesPerBlock(param);
	return false;
      }

    public:
      CopyCuda(Output &Y, Input &X, int length) : X(X), Y(Y), length(length) { ; }
      virtual ~CopyCuda() { ; }

      TuneKey tuneKey() const {
	std::stringstream vol, aux;
	vol << blasConstants.x[0] << "x";
	vol << blasConstants.x[1] << "x";
	vol << blasConstants.x[2] << "x";
	vol << blasConstants.x[3];
	aux << "stride=" << blasConstants.stride << ",out_prec=" << Y.Precision() << ",in_prec=" << X.Precision();
	return TuneKey(vol.str(), "copyKernel", aux.str());
      }  

      void apply(const hipStream_t &stream) {
	TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
	copyKernel<FloatN, N><<<tp.grid, tp.block, tp.shared_bytes, stream>>>(Y, X, length);
      }

      void preTune() { ; } // no need to save state for copy kernels
      void postTune() { ; } // no need to restore state for copy kernels

      long long flops() const { return 0; }
      long long bytes() const { 
	const int Ninternal = (sizeof(FloatN)/sizeof(((FloatN*)0)->x))*N;
	size_t bytes = (X.Precision() + Y.Precision())*Ninternal;
	if (X.Precision() == QUDA_HALF_PRECISION) bytes += sizeof(float);
	if (Y.Precision() == QUDA_HALF_PRECISION) bytes += sizeof(float);
	return bytes*length; 
      }
    };

    void copyCuda(cudaColorSpinorField &dst, const cudaColorSpinorField &src) {
      if (&src == &dst) return; // aliasing fields
      if (src.Nspin() != 1 && src.Nspin() != 4) errorQuda("nSpin(%d) not supported\n", src.Nspin());

      if (dst.SiteSubset() == QUDA_FULL_SITE_SUBSET || src.SiteSubset() == QUDA_FULL_SITE_SUBSET) {
	if (src.SiteSubset() != dst.SiteSubset()) 
	  errorQuda("Spinor fields do not have matching subsets dst=%d src=%d\n", 
		    dst.SiteSubset(), src.SiteSubset());
	copy::copyCuda(dst.Even(), src.Even());
	copy::copyCuda(dst.Odd(), src.Odd());
	return;
      }

      checkSpinorLength(dst, src);

      for (int d=0; d<QUDA_MAX_DIM; d++) blasConstants.x[d] = src.X()[d];
      blasConstants.stride = src.Stride();

      // For a given dst precision, there are two non-trivial possibilities for the
      // src precision.

      // FIXME: use traits to encapsulate register type for shorts -
      // will reduce template type parameters from 3 to 2

      blas_bytes += (unsigned long long)src.RealLength()*(src.Precision() + dst.Precision());
      
      if (dst.Precision() == src.Precision()) {
	if (src.Bytes() != dst.Bytes()) errorQuda("Precisions match, but bytes do not");
	hipMemcpy(dst.V(), src.V(), dst.Bytes(), hipMemcpyDeviceToDevice);
	if (dst.Precision() == QUDA_HALF_PRECISION) {
	  hipMemcpy(dst.Norm(), src.Norm(), dst.NormBytes(), hipMemcpyDeviceToDevice);
	  blas_bytes += 2*(unsigned long long)dst.RealLength()*sizeof(float);
	}
      } else if (dst.Precision() == QUDA_DOUBLE_PRECISION && src.Precision() == QUDA_SINGLE_PRECISION) {
	if (src.Nspin() == 4){
	  Spinor<float4, float4, float4, 6, 0, 0> src_tex(src);
	  Spinor<float4, float2, double2, 6, 1> dst_spinor(dst);
	  CopyCuda<float4, 6, Spinor<float4, float2, double2, 6, 1>, 
		   Spinor<float4, float4, float4, 6, 0, 0> >
	    copy(dst_spinor, src_tex, src.Volume());
	  copy.apply(*getBlasStream());	
      } else { //src.Nspin() == 1
	  Spinor<float2, float2, float2, 3, 0, 0> src_tex(src);
	  Spinor<float2, float2, double2, 3, 1> dst_spinor(dst);
	  CopyCuda<float2, 3, Spinor<float2, float2, double2, 3, 1>,
		   Spinor<float2, float2, float2, 3, 0, 0> >
	    copy(dst_spinor, src_tex, src.Volume());
	  copy.apply(*getBlasStream());	
    } 
  } else if (dst.Precision() == QUDA_SINGLE_PRECISION && src.Precision() == QUDA_DOUBLE_PRECISION) {
	if (src.Nspin() == 4){
	  Spinor<float4, float2, double2, 6, 0, 0> src_tex(src);
	  Spinor<float4, float4, float4, 6, 1> dst_spinor(dst);
	  CopyCuda<float4, 6, Spinor<float4, float4, float4, 6, 1>,
		   Spinor<float4, float2, double2, 6, 0, 0> >
	    copy(dst_spinor, src_tex, src.Volume());
	  copy.apply(*getBlasStream());	
      } else { //src.Nspin() ==1
	  Spinor<float2, float2, double2, 3, 0, 0> src_tex(src);
	  Spinor<float2, float2, float2, 3, 1> dst_spinor(dst);
	  CopyCuda<float2, 3, Spinor<float2, float2, float2, 3, 1>,
		   Spinor<float2, float2, double2, 3, 0, 0> >
	  copy(dst_spinor, src_tex, src.Volume());
	  copy.apply(*getBlasStream());	
      }
  } else if (dst.Precision() == QUDA_SINGLE_PRECISION && src.Precision() == QUDA_HALF_PRECISION) {
	blas_bytes += (unsigned long long)src.Volume()*sizeof(float);
	if (src.Nspin() == 4){      
	  Spinor<float4, float4, short4, 6, 0, 0> src_tex(src);
	  Spinor<float4, float4, float4, 6, 1> dst_spinor(dst);
	  CopyCuda<float4, 6, Spinor<float4, float4, float4, 6, 1>,
		   Spinor<float4, float4, short4, 6, 0, 0> >
	    copy(dst_spinor, src_tex, src.Volume());
	  copy.apply(*getBlasStream());	
      } else { //nSpin== 1;
	  Spinor<float2, float2, short2, 3, 0, 0> src_tex(src);
	  Spinor<float2, float2, float2, 3, 1> dst_spinor(dst);
	  CopyCuda<float2, 3, Spinor<float2, float2, float2, 3, 1>,
		   Spinor<float2, float2, short2, 3, 0, 0> >
	    copy(dst_spinor, src_tex, src.Volume());
	  copy.apply(*getBlasStream());	
    }
  } else if (dst.Precision() == QUDA_HALF_PRECISION && src.Precision() == QUDA_SINGLE_PRECISION) {
	blas_bytes += (unsigned long long)dst.Volume()*sizeof(float);
	if (src.Nspin() == 4){
	  Spinor<float4, float4, float4, 6, 0, 0> src_tex(src);
	  Spinor<float4, float4, short4, 6, 1> dst_spinor(dst);
	  CopyCuda<float4, 6, Spinor<float4, float4, short4, 6, 1>,
		   Spinor<float4, float4, float4, 6, 0, 0> >
	    copy(dst_spinor, src_tex, src.Volume());
	  copy.apply(*getBlasStream());	
      } else { //nSpin == 1
	  Spinor<float2, float2, float2, 3, 0, 0> src_tex(src);
	  Spinor<float2, float2, short2, 3, 1> dst_spinor(dst);
	  CopyCuda<float2, 3, Spinor<float2, float2, short2, 3, 1>,
		   Spinor<float2, float2, float2, 3, 0, 0> >
	  copy(dst_spinor, src_tex, src.Volume());
  copy.apply(*getBlasStream());	
}
  } else if (dst.Precision() == QUDA_DOUBLE_PRECISION && src.Precision() == QUDA_HALF_PRECISION) {
	blas_bytes += (unsigned long long)src.Volume()*sizeof(float);
	if (src.Nspin() == 4){
	  Spinor<double2, float4, short4, 12, 0, 0> src_tex(src);
	  Spinor<double2, double2, double2, 12, 1> dst_spinor(dst);
	  CopyCuda<double2, 12, Spinor<double2, double2, double2, 12, 1>,
		   Spinor<double2, float4, short4, 12, 0, 0> >
	    copy(dst_spinor, src_tex, src.Volume());
	  copy.apply(*getBlasStream());	
      } else { //nSpin == 1
	  Spinor<double2, float2, short2, 3, 0, 0> src_tex(src);
	  Spinor<double2, double2, double2, 3, 1> dst_spinor(dst);
	  CopyCuda<double2, 3, Spinor<double2, double2, double2, 3, 1>,
		   Spinor<double2, float2, short2, 3, 0, 0> >
	    copy(dst_spinor, src_tex, src.Volume());
	  copy.apply(*getBlasStream());	
    }
  } else if (dst.Precision() == QUDA_HALF_PRECISION && src.Precision() == QUDA_DOUBLE_PRECISION) {
	blas_bytes += (unsigned long long)dst.Volume()*sizeof(float);
	if (src.Nspin() == 4){
	  Spinor<double2, double2, double2, 12, 0, 0> src_tex(src);
	  Spinor<double2, double4, short4, 12, 1> dst_spinor(dst);
	  CopyCuda<double2, 12, Spinor<double2, double4, short4, 12, 1>,
		   Spinor<double2, double2, double2, 12, 0, 0> >
	    copy(dst_spinor, src_tex, src.Volume());
	  copy.apply(*getBlasStream());	
      } else { //nSpin == 1
	  Spinor<double2, double2, double2, 3, 0, 0> src_tex(src);
	  Spinor<double2, double2, short2, 3, 1> dst_spinor(dst);
	  CopyCuda<double2, 3, Spinor<double2, double2, short2, 3, 1>,
		   Spinor<double2, double2, double2, 3, 0, 0> >
	  copy(dst_spinor, src_tex, src.Volume());
  copy.apply(*getBlasStream());	
}
  } else {
	errorQuda("Invalid precision combination dst=%d and src=%d", dst.Precision(), src.Precision());
      }
      
      checkCudaError();
    }

  } // namespace copy

  void copyCuda(cudaColorSpinorField &dst, const cudaColorSpinorField &src) {
    copy::copyCuda(dst, src);
  }
  
} // namespace quda
