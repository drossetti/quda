#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <typeinfo>

#include <color_spinor_field.h>
#include <blas_quda.h>

#include <string.h>
#include <iostream>
#include <misc_helpers.h>
#include <face_quda.h>
#include <dslash_quda.h>

#ifdef DEVICE_PACK
#define REORDER_LOCATION QUDA_CUDA_FIELD_LOCATION
#else
#define REORDER_LOCATION QUDA_CPU_FIELD_LOCATION
#endif

int zeroCopy = 0;

namespace quda {

  int cudaColorSpinorField::initGhostFaceBuffer = 0;
  void* cudaColorSpinorField::ghostFaceBuffer; //gpu memory
  void* cudaColorSpinorField::fwdGhostFaceBuffer[QUDA_MAX_DIM]; //pointers to ghostFaceBuffer
  void* cudaColorSpinorField::backGhostFaceBuffer[QUDA_MAX_DIM]; //pointers to ghostFaceBuffer
  size_t cudaColorSpinorField::ghostFaceBytes = 0;

  /*cudaColorSpinorField::cudaColorSpinorField() : 
    ColorSpinorField(), v(0), norm(0), alloc(false), init(false) {

    }*/

  cudaColorSpinorField::cudaColorSpinorField(const ColorSpinorParam &param) : 
    ColorSpinorField(param), alloc(false), init(true), texInit(false), 
    initComms(false), nFaceComms(0) {

    // this must come before create
    if (param.create == QUDA_REFERENCE_FIELD_CREATE) {
      v = param.v;
      norm = param.norm;
    }

    create(param.create);

    if  (param.create == QUDA_NULL_FIELD_CREATE) {
      // do nothing
    } else if (param.create == QUDA_ZERO_FIELD_CREATE) {
      zero();
    } else if (param.create == QUDA_REFERENCE_FIELD_CREATE) {
      // dp nothing
    } else if (param.create == QUDA_COPY_FIELD_CREATE){
      errorQuda("not implemented");
    }
    checkCudaError();
  }

  cudaColorSpinorField::cudaColorSpinorField(const cudaColorSpinorField &src) : 
    ColorSpinorField(src), alloc(false), init(true), texInit(false), 
    initComms(false), nFaceComms(0) {
    create(QUDA_COPY_FIELD_CREATE);
    copySpinorField(src);
  }

  // creates a copy of src, any differences defined in param
  cudaColorSpinorField::cudaColorSpinorField(const ColorSpinorField &src, 
					     const ColorSpinorParam &param) :
    ColorSpinorField(src), alloc(false), init(true), texInit(false), 
    initComms(false), nFaceComms(0) {  

    // can only overide if we are not using a reference or parity special case
    if (param.create != QUDA_REFERENCE_FIELD_CREATE || 
	(param.create == QUDA_REFERENCE_FIELD_CREATE && 
	 src.SiteSubset() == QUDA_FULL_SITE_SUBSET && 
	 param.siteSubset == QUDA_PARITY_SITE_SUBSET && 
	 typeid(src) == typeid(cudaColorSpinorField) ) ) {
      reset(param);
    } else {
      errorQuda("Undefined behaviour"); // else silent bug possible?
    }

    // This must be set before create is called
    if (param.create == QUDA_REFERENCE_FIELD_CREATE) {
      if (typeid(src) == typeid(cudaColorSpinorField)) {
	v = (void*)src.V();
	norm = (void*)src.Norm();
      } else {
	errorQuda("Cannot reference a non-cuda field");
      }
    }

    create(param.create);

    if (param.create == QUDA_NULL_FIELD_CREATE) {
      // do nothing
    } else if (param.create == QUDA_ZERO_FIELD_CREATE) {
      zero();
    } else if (param.create == QUDA_COPY_FIELD_CREATE) {
      copySpinorField(src);
    } else if (param.create == QUDA_REFERENCE_FIELD_CREATE) {
      // do nothing
    } else {
      errorQuda("CreateType %d not implemented", param.create);
    }

    clearGhostPointers();
  }

  cudaColorSpinorField::cudaColorSpinorField(const ColorSpinorField &src) 
    : ColorSpinorField(src), alloc(false), init(true), texInit(false), 
      initComms(false), nFaceComms(0) {
    create(QUDA_COPY_FIELD_CREATE);
    copySpinorField(src);
    clearGhostPointers();
  }

  ColorSpinorField& cudaColorSpinorField::operator=(const ColorSpinorField &src) {
    if (typeid(src) == typeid(cudaColorSpinorField)) {
      *this = (dynamic_cast<const cudaColorSpinorField&>(src));
    } else if (typeid(src) == typeid(cpuColorSpinorField)) {
      *this = (dynamic_cast<const cpuColorSpinorField&>(src));
    } else {
      errorQuda("Unknown input ColorSpinorField %s", typeid(src).name());
    }
    return *this;
  }

  cudaColorSpinorField& cudaColorSpinorField::operator=(const cudaColorSpinorField &src) {
    if (&src != this) {
      // keep current attributes unless unset
      if (!ColorSpinorField::init) { // note this will turn a reference field into a regular field
	destroy();
	destroyComms(); // not sure if this necessary
	ColorSpinorField::operator=(src);
	create(QUDA_COPY_FIELD_CREATE);
      }
      copySpinorField(src);
    }
    return *this;
  }

  cudaColorSpinorField& cudaColorSpinorField::operator=(const cpuColorSpinorField &src) {
    // keep current attributes unless unset
    if (!ColorSpinorField::init) { // note this will turn a reference field into a regular field
      destroy();
      ColorSpinorField::operator=(src);
      create(QUDA_COPY_FIELD_CREATE);
    }
    loadSpinorField(src);
    return *this;
  }

  cudaColorSpinorField::~cudaColorSpinorField() {
    destroyComms();
    destroy();
  }

  bool cudaColorSpinorField::isNative() const {

    if (precision == QUDA_DOUBLE_PRECISION) {
      if (fieldOrder == QUDA_FLOAT2_FIELD_ORDER) return true;
    } else if (precision == QUDA_SINGLE_PRECISION) {
      if (nSpin == 4) {
	if (fieldOrder == QUDA_FLOAT4_FIELD_ORDER) return true;
      } else if (nSpin == 1) {
	if (fieldOrder == QUDA_FLOAT2_FIELD_ORDER) return true;
      }
    } else if (precision == QUDA_HALF_PRECISION) {
      if (nSpin == 4) {
	if (fieldOrder == QUDA_FLOAT4_FIELD_ORDER) return true;
      } else if (nSpin == 1) {
	if (fieldOrder == QUDA_FLOAT2_FIELD_ORDER) return true;
      }
    }

    return false;
  }

  void cudaColorSpinorField::create(const QudaFieldCreate create) {

    if (siteSubset == QUDA_FULL_SITE_SUBSET && siteOrder != QUDA_EVEN_ODD_SITE_ORDER) {
      errorQuda("Subset not implemented");
    }

    if (create != QUDA_REFERENCE_FIELD_CREATE) {
      v = device_malloc(bytes);
      if (precision == QUDA_HALF_PRECISION) {
	norm = device_malloc(norm_bytes);
      }
      alloc = true;
    }

    if (siteSubset == QUDA_FULL_SITE_SUBSET) {
      // create the associated even and odd subsets
      ColorSpinorParam param;
      param.siteSubset = QUDA_PARITY_SITE_SUBSET;
      param.nDim = nDim;
      memcpy(param.x, x, nDim*sizeof(int));
      param.x[0] /= 2; // set single parity dimensions
      param.create = QUDA_REFERENCE_FIELD_CREATE;
      param.v = v;
      param.norm = norm;
      even = new cudaColorSpinorField(*this, param);
      odd = new cudaColorSpinorField(*this, param);

      // need this hackery for the moment (need to locate the odd pointer half way into the full field)
      (dynamic_cast<cudaColorSpinorField*>(odd))->v = (void*)((char*)v + bytes/2);
      if (precision == QUDA_HALF_PRECISION) 
	(dynamic_cast<cudaColorSpinorField*>(odd))->norm = (void*)((char*)norm + norm_bytes/2);

#ifdef USE_TEXTURE_OBJECTS
      dynamic_cast<cudaColorSpinorField*>(even)->destroyTexObject();
      dynamic_cast<cudaColorSpinorField*>(even)->createTexObject();
      dynamic_cast<cudaColorSpinorField*>(odd)->destroyTexObject();
      dynamic_cast<cudaColorSpinorField*>(odd)->createTexObject();
#endif
    }

    if (create != QUDA_REFERENCE_FIELD_CREATE) {
      if (siteSubset != QUDA_FULL_SITE_SUBSET) {
	zeroPad();
      } else {
	(dynamic_cast<cudaColorSpinorField*>(even))->zeroPad();
	(dynamic_cast<cudaColorSpinorField*>(odd))->zeroPad();
      }
    }

#ifdef USE_TEXTURE_OBJECTS
    createTexObject();
#endif

    checkCudaError();
  }

#ifdef USE_TEXTURE_OBJECTS
  void cudaColorSpinorField::createTexObject() {

    if (isNative()) {
      if (texInit) errorQuda("Already bound textures");
      
      // create the texture for the field components
      
      hipChannelFormatDesc desc;
      memset(&desc, 0, sizeof(hipChannelFormatDesc));
      if (precision == QUDA_SINGLE_PRECISION) desc.f = hipChannelFormatKindFloat;
      else desc.f = hipChannelFormatKindSigned; // half is short, double is int2
      
      // staggered fields in half and single are always two component
      if (nSpin == 1 && (precision == QUDA_HALF_PRECISION || precision == QUDA_SINGLE_PRECISION)) {
	desc.x = 8*precision;
	desc.y = 8*precision;
	desc.z = 0;
	desc.w = 0;
      } else { // all others are four component
	desc.x = (precision == QUDA_DOUBLE_PRECISION) ? 32 : 8*precision;
	desc.y = (precision == QUDA_DOUBLE_PRECISION) ? 32 : 8*precision;
	desc.z = (precision == QUDA_DOUBLE_PRECISION) ? 32 : 8*precision;
	desc.w = (precision == QUDA_DOUBLE_PRECISION) ? 32 : 8*precision;
      }
      
      hipResourceDesc resDesc;
      memset(&resDesc, 0, sizeof(resDesc));
      resDesc.resType = hipResourceTypeLinear;
      resDesc.res.linear.devPtr = v;
      resDesc.res.linear.desc = desc;
      resDesc.res.linear.sizeInBytes = bytes;
      
      hipTextureDesc texDesc;
      memset(&texDesc, 0, sizeof(texDesc));
      if (precision == QUDA_HALF_PRECISION) texDesc.readMode = hipReadModeNormalizedFloat;
      else texDesc.readMode = hipReadModeElementType;
      
      hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL);
      checkCudaError();
      
      // create the texture for the norm components
      if (precision == QUDA_HALF_PRECISION) {
	hipChannelFormatDesc desc;
	memset(&desc, 0, sizeof(hipChannelFormatDesc));
	desc.f = hipChannelFormatKindFloat;
	desc.x = 8*QUDA_SINGLE_PRECISION; desc.y = 0; desc.z = 0; desc.w = 0;
	
	hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeLinear;
	resDesc.res.linear.devPtr = norm;
	resDesc.res.linear.desc = desc;
	resDesc.res.linear.sizeInBytes = norm_bytes;
	
	hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.readMode = hipReadModeElementType;
	
	hipCreateTextureObject(&texNorm, &resDesc, &texDesc, NULL);
	checkCudaError();
      }
      
      texInit = true;
    }
  }

  void cudaColorSpinorField::destroyTexObject() {
    if (isNative() && texInit) {
      hipDestroyTextureObject(tex);
      if (precision == QUDA_HALF_PRECISION) hipDestroyTextureObject(texNorm);
      texInit = false;
      checkCudaError();
    }
  }
#endif

  void cudaColorSpinorField::destroy() {
    if (alloc) {
      device_free(v);
      if (precision == QUDA_HALF_PRECISION) device_free(norm);
      if (siteSubset == QUDA_FULL_SITE_SUBSET) {
	delete even;
	delete odd;
      }
      alloc = false;
    }

#ifdef USE_TEXTURE_OBJECTS
    destroyTexObject();
#endif

  }

  cudaColorSpinorField& cudaColorSpinorField::Even() const { 
    if (siteSubset == QUDA_FULL_SITE_SUBSET) {
      return *(dynamic_cast<cudaColorSpinorField*>(even)); 
    }

    errorQuda("Cannot return even subset of %d subset", siteSubset);
    exit(-1);
  }

  cudaColorSpinorField& cudaColorSpinorField::Odd() const {
    if (siteSubset == QUDA_FULL_SITE_SUBSET) {
      return *(dynamic_cast<cudaColorSpinorField*>(odd)); 
    }

    errorQuda("Cannot return odd subset of %d subset", siteSubset);
    exit(-1);
  }

  // cuda's floating point format, IEEE-754, represents the floating point
  // zero as 4 zero bytes
  void cudaColorSpinorField::zero() {
    hipMemsetAsync(v, 0, bytes, streams[Nstream-1]);
    if (precision == QUDA_HALF_PRECISION) hipMemsetAsync(norm, 0, norm_bytes, streams[Nstream-1]);
  }


  void cudaColorSpinorField::zeroPad() {
    size_t pad_bytes = (stride - volume) * precision * fieldOrder;
    int Npad = nColor * nSpin * 2 / fieldOrder;
    for (int i=0; i<Npad; i++) {
      if (pad_bytes) hipMemset((char*)v + (volume + i*stride)*fieldOrder*precision, 0, pad_bytes);
    }
  }

  void cudaColorSpinorField::copy(const cudaColorSpinorField &src) {
    checkField(*this, src);
    copyCuda(*this, src);
  }

  void cudaColorSpinorField::copySpinorField(const ColorSpinorField &src) {
    
    // src is on the device and is native
    if (typeid(src) == typeid(cudaColorSpinorField) && 
	isNative() && dynamic_cast<const cudaColorSpinorField &>(src).isNative()) {
      copy(dynamic_cast<const cudaColorSpinorField&>(src));
    } else if (typeid(src) == typeid(cudaColorSpinorField)) {
      copyGenericColorSpinor(*this, src, QUDA_CUDA_FIELD_LOCATION);
    } else if (typeid(src) == typeid(cpuColorSpinorField)) { // src is on the host
      loadSpinorField(src);
    } else {
      errorQuda("Unknown input ColorSpinorField %s", typeid(src).name());
    }
  } 

  void cudaColorSpinorField::loadSpinorField(const ColorSpinorField &src) {

    if (REORDER_LOCATION == QUDA_CPU_FIELD_LOCATION && 
	typeid(src) == typeid(cpuColorSpinorField)) {
      resizeBufferPinned(bytes + norm_bytes);
      memset(bufferPinned, 0, bytes+norm_bytes); // FIXME (temporary?) bug fix for padding

      copyGenericColorSpinor(*this, src, QUDA_CPU_FIELD_LOCATION, 
			     bufferPinned, 0, (char*)bufferPinned+bytes, 0);

      hipMemcpy(v, bufferPinned, bytes, hipMemcpyHostToDevice);
      hipMemcpy(norm, (char*)bufferPinned+bytes, norm_bytes, hipMemcpyHostToDevice);
    } else if (typeid(src) == typeid(cudaColorSpinorField)) {
      copyGenericColorSpinor(*this, src, QUDA_CUDA_FIELD_LOCATION);
    } else {
      void *Src, *srcNorm;
      if (!zeroCopy) {
	resizeBufferDevice(src.Bytes()+src.NormBytes());
	Src = bufferDevice;
	srcNorm = (char*)bufferDevice + src.Bytes();	
	hipMemcpy(Src, src.V(), src.Bytes(), hipMemcpyHostToDevice);
	hipMemcpy(srcNorm, src.Norm(), src.NormBytes(), hipMemcpyHostToDevice);
      } else {
	resizeBufferPinned(src.Bytes()+src.NormBytes());
	memcpy(bufferPinned, src.V(), src.Bytes());
	memcpy((char*)bufferPinned+src.Bytes(), src.Norm(), src.NormBytes());

	hipHostGetDevicePointer(&Src, bufferPinned, 0);
	srcNorm = (void*)((char*)Src + src.Bytes());
      }

      hipMemset(v, 0, bytes); // FIXME (temporary?) bug fix for padding
      copyGenericColorSpinor(*this, src, QUDA_CUDA_FIELD_LOCATION, 0, Src, 0, srcNorm);
    }

    checkCudaError();
    return;
  }


  void cudaColorSpinorField::saveSpinorField(ColorSpinorField &dest) const {

    if (REORDER_LOCATION == QUDA_CPU_FIELD_LOCATION && 
	typeid(dest) == typeid(cpuColorSpinorField)) {
      resizeBufferPinned(bytes+norm_bytes);
      hipMemcpy(bufferPinned, v, bytes, hipMemcpyDeviceToHost);
      hipMemcpy((char*)bufferPinned+bytes, norm, norm_bytes, hipMemcpyDeviceToHost);

      copyGenericColorSpinor(dest, *this, QUDA_CPU_FIELD_LOCATION, 
			     0, bufferPinned, 0, (char*)bufferPinned+bytes);
    } else if (typeid(dest) == typeid(cudaColorSpinorField)) {
      copyGenericColorSpinor(dest, *this, QUDA_CUDA_FIELD_LOCATION);
    } else {
      void *dst, *dstNorm;
      if (!zeroCopy) {
	resizeBufferDevice(dest.Bytes()+dest.NormBytes());
	dst = bufferDevice;
	dstNorm = (char*)bufferDevice+dest.Bytes();
      } else {
	resizeBufferPinned(dest.Bytes()+dest.NormBytes());
	hipHostGetDevicePointer(&dst, bufferPinned, 0);
	dstNorm = (char*)dst+dest.Bytes();
      }
      copyGenericColorSpinor(dest, *this, QUDA_CUDA_FIELD_LOCATION, dst, v, dstNorm, 0);

      if (!zeroCopy) {
	hipMemcpy(dest.V(), dst, dest.Bytes(), hipMemcpyDeviceToHost);
	hipMemcpy(dest.Norm(), dstNorm, dest.NormBytes(), hipMemcpyDeviceToHost);
      } else {
	memcpy(dest.V(), bufferPinned, dest.Bytes());
	memcpy(dest.Norm(), (char*)bufferPinned+dest.Bytes(), dest.NormBytes());
      }
    }

    checkCudaError();
    return;
  }

  void cudaColorSpinorField::allocateGhostBuffer(int nFace) {
    int Nint = nColor * nSpin * 2; // number of internal degrees of freedom
    if (nSpin == 4) Nint /= 2; // spin projection for Wilson

    // compute size of buffer required
    size_t faceBytes = 0;
    for (int i=0; i<4; i++) {
      if(!commDimPartitioned(i)) continue;
      faceBytes += 2*nFace*ghostFace[i]*Nint*precision;
      // add extra space for the norms for half precision
      if (precision == QUDA_HALF_PRECISION) faceBytes += 2*nFace*ghostFace[i]*sizeof(float);
    }

    // only allocate if not already allocated or buffer required is bigger than previously
    if(initGhostFaceBuffer == 0 || faceBytes > ghostFaceBytes){    

      if (initGhostFaceBuffer) device_free(ghostFaceBuffer); 

      if (faceBytes > 0) {
	ghostFaceBuffer = device_malloc(faceBytes);
	initGhostFaceBuffer = 1;
	ghostFaceBytes = faceBytes;
      }

    }

    size_t offset = 0;
    for (int i=0; i<4; i++) {
      if(!commDimPartitioned(i)) continue;
      
      backGhostFaceBuffer[i] = (void*)(((char*)ghostFaceBuffer) + offset);
      offset += nFace*ghostFace[i]*Nint*precision;
      if (precision == QUDA_HALF_PRECISION) offset += nFace*ghostFace[i]*sizeof(float);
      
      fwdGhostFaceBuffer[i] = (void*)(((char*)ghostFaceBuffer) + offset);
      offset += nFace*ghostFace[i]*Nint*precision;
      if (precision == QUDA_HALF_PRECISION) offset += nFace*ghostFace[i]*sizeof(float);
    }   
    
  }


  void cudaColorSpinorField::freeGhostBuffer(void)
  {
    if (!initGhostFaceBuffer) return;
  
    device_free(ghostFaceBuffer); 

    for(int i=0;i < 4; i++){
      if(!commDimPartitioned(i)) continue;
      backGhostFaceBuffer[i] = NULL;
      fwdGhostFaceBuffer[i] = NULL;
    }
    initGhostFaceBuffer = 0;  
  }

  // pack the ghost zone into a contiguous buffer for communications
  void cudaColorSpinorField::packGhost(FullClover &clov, FullClover &clovInv,
				       const int nFace, const QudaParity parity, 
                                       const int dim, const QudaDirection dir,
				       const int dagger, hipStream_t *stream, 
				       void *buffer, double a) 
  {
    int face_num;
    if(dir == QUDA_BACKWARDS){
      face_num = 0;
    }else if(dir == QUDA_FORWARDS){
      face_num = 1;
    }else{
      face_num = 2;
    }
#ifdef MULTI_GPU
    void *packBuffer = buffer ? buffer : ghostFaceBuffer;
    packFace(packBuffer, *this, clov, clovInv, nFace, dagger, parity, dim, face_num, *stream, a); 
#else
    errorQuda("packGhost not built on single-GPU build");
#endif

  }
 
  // pack the ghost zone into a contiguous buffer for communications
  void cudaColorSpinorField::packGhost(const int nFace, const QudaParity parity, 
                                       const int dim, const QudaDirection dir,
				       const int dagger, hipStream_t *stream, 
				       void *buffer, double a, double b) 
  {
    int face_num;
    if(dir == QUDA_BACKWARDS){
      face_num = 0;
    }else if(dir == QUDA_FORWARDS){
      face_num = 1;
    }else{
      face_num = 2;
    }
#ifdef MULTI_GPU
    void *packBuffer = buffer ? buffer : ghostFaceBuffer;
    packFace(packBuffer, *this, nFace, dagger, parity, dim, face_num, *stream, a, b); 
#else
    errorQuda("packGhost not built on single-GPU build");
#endif

  }
 
  // send the ghost zone to the host
  void cudaColorSpinorField::sendGhost(void *ghost_spinor, const int nFace, const int dim, 
				       const QudaDirection dir, const int dagger, 
				       hipStream_t *stream) {

#ifdef MULTI_GPU
    int Nvec = (nSpin == 1 || precision == QUDA_DOUBLE_PRECISION) ? 2 : 4;
    int Nint = (nColor * nSpin * 2) / (nSpin == 4 ? 2 : 1);  // (spin proj.) degrees of freedom
    
    if (dim !=3 || getKernelPackT() || getTwistPack()) { // use kernels to pack into contiguous buffers then a single hipMemcpy

      size_t bytes = nFace*Nint*ghostFace[dim]*precision;
      if (precision == QUDA_HALF_PRECISION) bytes += nFace*ghostFace[dim]*sizeof(float);
      void* gpu_buf = 
	(dir == QUDA_BACKWARDS) ? this->backGhostFaceBuffer[dim] : this->fwdGhostFaceBuffer[dim];

      hipMemcpyAsync(ghost_spinor, gpu_buf, bytes, hipMemcpyDeviceToHost, *stream); 
    } else if(this->TwistFlavor() != QUDA_TWIST_NONDEG_DOUBLET){ // do multiple cudaMemcpys

      int Npad = Nint / Nvec; // number Nvec buffers we have
      int Nt_minus1_offset = (volume - nFace*ghostFace[3]); // N_t -1 = Vh-Vsh
      int offset = 0;
      if (nSpin == 1) {
	offset = (dir == QUDA_BACKWARDS) ? 0 : Nt_minus1_offset;
      } else if (nSpin == 4) {    
	// !dagger: send lower components backwards, send upper components forwards
	// dagger: send upper components backwards, send lower components forwards
	bool upper = dagger ? true : false; // Fwd is !Back  
	if (dir == QUDA_FORWARDS) upper = !upper;
	int lower_spin_offset = Npad*stride;
	if (upper) offset = (dir == QUDA_BACKWARDS ? 0 : Nt_minus1_offset);
	else offset = lower_spin_offset + (dir == QUDA_BACKWARDS ? 0 : Nt_minus1_offset);
      }
    
      // QUDA Memcpy NPad's worth. 
      //  -- Dest will point to the right beginning PAD. 
      //  -- Each Pad has size Nvec*Vsh Floats. 
      //  --  There is Nvec*Stride Floats from the start of one PAD to the start of the next

      void *dst = (char*)ghost_spinor;
      void *src = (char*)v + offset*Nvec*precision;
      size_t len = nFace*ghostFace[3]*Nvec*precision;     
      size_t spitch = stride*Nvec*precision;
      hipMemcpy2DAsync(dst, len, src, spitch, len, Npad, hipMemcpyDeviceToHost, *stream);

      if (precision == QUDA_HALF_PRECISION) {
	int norm_offset = (dir == QUDA_BACKWARDS) ? 0 : Nt_minus1_offset*sizeof(float);
	void *dst = (char*)ghost_spinor + nFace*Nint*ghostFace[3]*precision;
	void *src = (char*)norm + norm_offset;
	hipMemcpyAsync(dst, src, nFace*ghostFace[3]*sizeof(float), hipMemcpyDeviceToHost, *stream); 
      }
    }else{
      int flavorVolume = volume / 2;
      int flavorTFace  = ghostFace[3] / 2;
      int Npad = Nint / Nvec; // number Nvec buffers we have
      int flavor1_Nt_minus1_offset = (flavorVolume - flavorTFace);
      int flavor2_Nt_minus1_offset = (volume - flavorTFace);
      int flavor1_offset = 0;
      int flavor2_offset = 0;
      // !dagger: send lower components backwards, send upper components forwards
      // dagger: send upper components backwards, send lower components forwards
      bool upper = dagger ? true : false; // Fwd is !Back
      if (dir == QUDA_FORWARDS) upper = !upper;
      int lower_spin_offset = Npad*stride;//ndeg tm: stride=2*flavor_volume+pad
      if (upper){
        flavor1_offset = (dir == QUDA_BACKWARDS ? 0 : flavor1_Nt_minus1_offset);
        flavor2_offset = (dir == QUDA_BACKWARDS ? flavorVolume : flavor2_Nt_minus1_offset);
      }else{
        flavor1_offset = lower_spin_offset + (dir == QUDA_BACKWARDS ? 0 : flavor1_Nt_minus1_offset);
        flavor2_offset = lower_spin_offset + (dir == QUDA_BACKWARDS ? flavorVolume : flavor2_Nt_minus1_offset);
      }

      // QUDA Memcpy NPad's worth.
      //  -- Dest will point to the right beginning PAD.
      //  -- Each Pad has size Nvec*Vsh Floats.
      //  --  There is Nvec*Stride Floats from the start of one PAD to the start of the next

      void *dst = (char*)ghost_spinor;
      void *src = (char*)v + flavor1_offset*Nvec*precision;
      size_t len = flavorTFace*Nvec*precision;
      size_t spitch = stride*Nvec*precision;//ndeg tm: stride=2*flavor_volume+pad
      size_t dpitch = 2*len;
      hipMemcpy2DAsync(dst, dpitch, src, spitch, len, Npad, hipMemcpyDeviceToHost, *stream);
      dst = (char*)ghost_spinor+len;
      src = (char*)v + flavor2_offset*Nvec*precision;
      hipMemcpy2DAsync(dst, dpitch, src, spitch, len, Npad, hipMemcpyDeviceToHost, *stream);

      if (precision == QUDA_HALF_PRECISION) {
        int Nt_minus1_offset = (flavorVolume - flavorTFace);
        int norm_offset = (dir == QUDA_BACKWARDS) ? 0 : Nt_minus1_offset*sizeof(float);
	void *dst = (char*)ghost_spinor + Nint*ghostFace[3]*precision;
	void *src = (char*)norm + norm_offset;
        size_t dpitch = flavorTFace*sizeof(float);
        size_t spitch = flavorVolume*sizeof(float);
	hipMemcpy2DAsync(dst, dpitch, src, spitch, flavorTFace*sizeof(float), 2, hipMemcpyDeviceToHost, *stream);
      }
    }
#else
    errorQuda("sendGhost not built on single-GPU build");
#endif

  }


  void cudaColorSpinorField::unpackGhost(const void* ghost_spinor, const int nFace, 
					 const int dim, const QudaDirection dir, 
					 const int dagger, hipStream_t* stream) 
  {
    int Nint = (nColor * nSpin * 2) / (nSpin == 4 ? 2 : 1);  // (spin proj.) degrees of freedom

    int len = nFace*ghostFace[dim]*Nint;
    int offset = length + ghostOffset[dim]*nColor*nSpin*2;
    offset += (dir == QUDA_BACKWARDS) ? 0 : len;

    void *dst = (char*)v + precision*offset;
    const void *src = ghost_spinor;

    hipMemcpyAsync(dst, src, len*precision, hipMemcpyHostToDevice, *stream);
    
    if (precision == QUDA_HALF_PRECISION) {
      // norm region of host ghost zone is at the end of the ghost_spinor

      int normlen = nFace*ghostFace[dim];
      int norm_offset = stride + ghostNormOffset[dim];
      norm_offset += (dir == QUDA_BACKWARDS) ? 0 : normlen;

      void *dst = static_cast<char*>(norm) + norm_offset*sizeof(float);
      const void *src = static_cast<const char*>(ghost_spinor)+nFace*Nint*ghostFace[dim]*precision; 
      hipMemcpyAsync(dst, src, normlen*sizeof(float), hipMemcpyHostToDevice, *stream);
    }

  }

  hipStream_t *stream;

  void cudaColorSpinorField::createComms(int nFace) {
    if (!initComms || nFaceComms != nFace) {

      // if we are requesting a new number of faces destroy and start over
      if(nFace != nFaceComms) destroyComms();

      if (siteSubset != QUDA_PARITY_SITE_SUBSET) 
	errorQuda("Only supports single parity fields");

#ifdef GPU_COMMS
      bool comms = false;
      for (int i=0; i<nDimComms; i++) if (commDimPartitioned(i)) comms = true;
      if (comms && precision == QUDA_HALF_PRECISION)
	errorQuda("GPU-aware communication not yet supported with half precision");
#endif

      if (nFace > maxNface) 
	errorQuda("Requested number of faces %d in communicator is greater than supported %d",
		  nFace, maxNface);

      // faceBytes is the sum of all face sizes 
      size_t faceBytes = 0;
      
      // nbytes is the size in bytes of each face
      size_t nbytes[QUDA_MAX_DIM];
      
      // The number of degrees of freedom per site for the given
      // field.  Currently assumes spin projection of a Wilson-like
      // field (so half the number of degrees of freedom).
      int Ndof = (2 * nSpin * nColor) / (nSpin==4 ? 2 : 1);

      for (int i=0; i<nDimComms; i++) {
	nbytes[i] = maxNface*surfaceCB[i]*Ndof*precision;
	if (precision == QUDA_HALF_PRECISION) nbytes[i] += maxNface*surfaceCB[i]*sizeof(float);
	if (siteSubset == QUDA_PARITY_SITE_SUBSET && i==0) nbytes[i] /= 2;
	if (!commDimPartitioned(i)) continue;
	faceBytes += 2*nbytes[i];
      }
      
      // use static pinned memory for face buffers
      resizeBufferPinned(2*faceBytes); // oversizes for GPU_COMMS case

      my_face = bufferPinned;
      from_face = static_cast<char*>(bufferPinned) + faceBytes;

      // assign pointers for each face - it's ok to alias for different Nface parameters
#ifndef GPU_COMMS
      size_t offset = 0;
#endif
      for (int i=0; i<nDimComms; i++) {
	if (!commDimPartitioned(i)) continue;
	
#ifdef GPU_COMMS
	size_t offset2 = precision*(length + ghostOffset[i]*nColor*nSpin*2);
	my_back_face[i] = backGhostFaceBuffer[i];
	from_back_face[i] = static_cast<char*>(v) + offset2;
#else
	my_back_face[i] = static_cast<char*>(my_face) + offset;
	from_back_face[i] = static_cast<char*>(from_face) + offset;
	offset += nbytes[i];
#endif
	
#ifdef GPU_COMMS
	offset2 += nFace*ghostFace[i]*Ndof*precision;
	my_fwd_face[i] = fwdGhostFaceBuffer[i];	
	from_fwd_face[i] = /*static_cast<char*>(ghost[i]) + nFace*ghostFace[i]*Ndof*precision; //*/static_cast<char*>(v) + offset2;
#else
	my_fwd_face[i] = static_cast<char*>(my_face) + offset;
	from_fwd_face[i] = static_cast<char*>(from_face) + offset;
	offset += nbytes[i];
#endif

      }

      // create a different message handler for each direction and Nface
      mh_send_fwd = new MsgHandle**[maxNface];
      mh_send_back = new MsgHandle**[maxNface];
      mh_recv_fwd = new MsgHandle**[maxNface];
      mh_recv_back = new MsgHandle**[maxNface];
      for (int j=0; j<maxNface; j++) {
	mh_send_fwd[j] = new MsgHandle*[2*nDimComms];
	mh_send_back[j] = new MsgHandle*[2*nDimComms];
	mh_recv_fwd[j] = new MsgHandle*[nDimComms];
	mh_recv_back[j] = new MsgHandle*[nDimComms];

	for (int i=0; i<nDimComms; i++) {
	  size_t nbytes_Nface = (nbytes[i] / maxNface) * (j+1);
	  if (!commDimPartitioned(i)) continue;
#ifdef GPU_COMMS
	  if (i != 3 || getKernelPackT()) {
#endif
	    mh_send_fwd[j][2*i+0] = comm_declare_send_relative(my_fwd_face[i], i, +1, nbytes_Nface);
	    mh_send_back[j][2*i+0] = comm_declare_send_relative(my_back_face[i], i, -1, nbytes_Nface);
	    mh_send_fwd[j][2*i+1] = mh_send_fwd[j][2*i]; // alias pointers
	    mh_send_back[j][2*i+1] = mh_send_back[j][2*i]; // alias pointers
#ifdef GPU_COMMS
	  } else { 
	    /* 
	       use a strided communicator, here we can't really use
	       the previously declared my_fwd_face and my_back_face
	       pointers since they don't really map 1-to-1 so let's
	       just compute the required base pointers and pass these
	       directly into the communicator construction
	    */
	    
	    int Nblocks = Ndof / Nvec(); // number of Nvec buffers we have
	    // start of last time slice chunk we are sending forwards
	    int endOffset = (volume - (j+1)*ghostFace[i]); 

	    size_t offset[4];
	    void *base[4];
	    if (nSpin == 1) { // staggered is invariant with dagger
	      offset[2*0 + 0] = 0;
	      offset[2*1 + 0] = endOffset;
	      offset[2*0 + 1] = offset[2*0 + 0];
	      offset[2*1 + 1] = offset[2*1 + 0];
	    } else if (nSpin == 4) {    
	      // !dagger: send last components backwards, send first components forwards
	      offset[2*0 + 0] = Nblocks*stride;
	      offset[2*1 + 0] = endOffset;
	      //  dagger: send first components backwards, send last components forwards
	      offset[2*0 + 1] = 0;
	      offset[2*1 + 1] = Nblocks*stride + endOffset;
	    } else {
	      errorQuda("Unsupported number of spin components");
	    }

	    for (int k=0; k<4; k++) {
	      base[k] = static_cast<char*>(v) + offset[k]*Nvec()*precision; // total offset in bytes
	    }

	    size_t blksize  = (j+1)*ghostFace[i]*Nvec()*precision; // (j+1) is number of faces
	    size_t Stride = stride*Nvec()*precision;

	    if (blksize * Nblocks != nbytes_Nface) 
	      errorQuda("Total strided message size does not match expected size");

	    //printf("\n%d strided sends with Nface=%d Nblocks=%d blksize=%d Stride=%d,
	    //	   i, j+1, Nblocks, blksize, Stride);

	    mh_send_fwd[j][2*i+0] = comm_declare_strided_send_relative(base[2], i, +1, 
								       blksize, Nblocks, Stride);
	    mh_send_back[j][2*i+0] = comm_declare_strided_send_relative(base[0], i, -1, 
									blksize, Nblocks, Stride);
	    if (nSpin ==4) { // dagger communicators
	      mh_send_fwd[j][2*i+1] = comm_declare_strided_send_relative(base[3], i, +1, 
									 blksize, Nblocks, Stride);
	      mh_send_back[j][2*i+1] = comm_declare_strided_send_relative(base[1], i, -1, 
									  blksize, Nblocks, Stride);
	    } else {
	      mh_send_fwd[j][2*i+1] = mh_send_fwd[j][2*i+0];
	      mh_send_back[j][2*i+1] = mh_send_back[j][2*i+0];
	    }
	  }
#endif // GPU_COMMS

	  mh_recv_fwd[j][i] = comm_declare_receive_relative(from_fwd_face[i], i, +1, nbytes_Nface);
	  mh_recv_back[j][i] = comm_declare_receive_relative(from_back_face[i], i, -1, nbytes_Nface);

	} // loop over dimension
      }
      
      initComms = true;
      nFaceComms = nFace;
    }
    checkCudaError();
  }
    
  void cudaColorSpinorField::destroyComms() {
    if (initComms) {
      for (int j=0; j<maxNface; j++) {
	for (int i=0; i<nDimComms; i++) {
	  if (commDimPartitioned(i)) {
	    comm_free(mh_recv_fwd[j][i]);
	    comm_free(mh_recv_back[j][i]);
	    comm_free(mh_send_fwd[j][2*i]);
	    comm_free(mh_send_back[j][2*i]);
	    // only in a special case are these not aliasing pointers
#ifdef GPU_COMMS
	    if (i == 3 && !getKernelPackT() && nSpin == 4) {
	      comm_free(mh_send_fwd[j][2*i+1]);
	      comm_free(mh_send_back[j][2*i+1]);
	    }
#endif // GPU_COMMS
	  }
	}
	delete []mh_recv_fwd[j];
	delete []mh_recv_back[j];
	delete []mh_send_fwd[j];
	delete []mh_send_back[j];
      }    
      delete []mh_recv_fwd;
      delete []mh_recv_back;
      delete []mh_send_fwd;
      delete []mh_send_back;
      
      for (int i=0; i<nDimComms; i++) {
	my_fwd_face[i] = NULL;
	my_back_face[i] = NULL;
	from_fwd_face[i] = NULL;
	from_back_face[i] = NULL;      
      }
      
      initComms = false;
      checkCudaError();
    }
  }

  void cudaColorSpinorField::pack(FullClover &clov, FullClover &clovInv, int nFace, int parity,
				  int dagger, hipStream_t *stream_p, bool zeroCopyPack, double a) {
    allocateGhostBuffer(nFace);   // allocate the ghost buffer if not yet allocated  
    createComms(nFace); // must call this first

    stream = stream_p;
    
    const int dim=-1; // pack all partitioned dimensions
 
    if (zeroCopyPack) {
      void *my_face_d;
      hipHostGetDevicePointer(&my_face_d, my_face, 0); // set the matching device pointer
      packGhost(clov, clovInv, nFace, (QudaParity)parity, dim, QUDA_BOTH_DIRS, dagger, &stream[0], my_face_d, a);
    } else {
      packGhost(clov, clovInv, nFace, (QudaParity)parity, dim, QUDA_BOTH_DIRS, dagger,  &stream[Nstream-1], 0, a);
    }
  }

  void cudaColorSpinorField::pack(int nFace, int parity, int dagger, hipStream_t *stream_p, 
				  bool zeroCopyPack, double a, double b) {
    allocateGhostBuffer(nFace);   // allocate the ghost buffer if not yet allocated  
    createComms(nFace); // must call this first

    stream = stream_p;
    
    const int dim=-1; // pack all partitioned dimensions
 
    if (zeroCopyPack) {
      void *my_face_d;
      hipHostGetDevicePointer(&my_face_d, my_face, 0); // set the matching device pointer
      packGhost(nFace, (QudaParity)parity, dim, QUDA_BOTH_DIRS, dagger, &stream[0], my_face_d, a, b);
    } else {
      packGhost(nFace, (QudaParity)parity, dim, QUDA_BOTH_DIRS, dagger,  &stream[Nstream-1], 0, a, b);
    }
  }

  void cudaColorSpinorField::gather(int nFace, int dagger, int dir) {
    int dim = dir/2;
    if(!commDimPartitioned(dim)) return;

    if (dir%2==0) {
      // backwards copy to host
      sendGhost(my_back_face[dim], nFace, dim, QUDA_BACKWARDS, dagger, &stream[2*dim+0]); 
    } else {
      // forwards copy to host
      sendGhost(my_fwd_face[dim], nFace, dim, QUDA_FORWARDS, dagger, &stream[2*dim+1]);
    }
  }

  void cudaColorSpinorField::commsStart(int nFace, int dir, int dagger) {
    int dim = dir / 2;
    if(!commDimPartitioned(dim)) return;
    
    if (dir%2 == 0) { // sending backwards
      // Prepost receive
      comm_start(mh_recv_fwd[nFace-1][dim]);
      comm_start(mh_send_back[nFace-1][2*dim+dagger]);
    } else { //sending forwards
      // Prepost receive
      comm_start(mh_recv_back[nFace-1][dim]);
      // Begin forward send
      comm_start(mh_send_fwd[nFace-1][2*dim+dagger]);
    }
  }

  int cudaColorSpinorField::commsQuery(int nFace, int dir, int dagger) {
    int dim = dir / 2;
    if(!commDimPartitioned(dim)) return 0;
    
    if(dir%2==0) {
      if (comm_query(mh_recv_fwd[nFace-1][dim]) && 
	  comm_query(mh_send_back[nFace-1][2*dim+dagger])) return 1;
    } else {
      if (comm_query(mh_recv_back[nFace-1][dim]) && 
	  comm_query(mh_send_fwd[nFace-1][2*dim+dagger])) return 1;
    }
    
    return 0;
  }

  void cudaColorSpinorField::scatter(int nFace, int dagger, int dir)
  {
    int dim = dir/2;
    if(!commDimPartitioned(dim)) return;
    
    // both scattering occurances now go through the same stream
    if (dir%2==0) {// receive from forwards
      unpackGhost(from_fwd_face[dim], nFace, dim, QUDA_FORWARDS, dagger, &stream[2*dim/*+0*/]);
    } else { // receive from backwards
      unpackGhost(from_back_face[dim], nFace, dim, QUDA_BACKWARDS, dagger, &stream[2*dim/*+1*/]);
    }
  }

  // Return the location of the field
  QudaFieldLocation cudaColorSpinorField::Location() const { return QUDA_CUDA_FIELD_LOCATION; }

  std::ostream& operator<<(std::ostream &out, const cudaColorSpinorField &a) {
    out << (const ColorSpinorField&)a;
    out << "v = " << a.v << std::endl;
    out << "norm = " << a.norm << std::endl;
    out << "alloc = " << a.alloc << std::endl;
    out << "init = " << a.init << std::endl;
    return out;
  }

} // namespace quda
