#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <typeinfo>

#include <color_spinor_field.h>
#include <blas_quda.h>

#include <string.h>
#include <iostream>
#include <misc_helpers.h>
#include <face_quda.h>
#include <dslash_quda.h>

#ifdef DEVICE_PACK
static const QudaFieldLocation reorder_location_ = QUDA_CUDA_FIELD_LOCATION;
#else
static const QudaFieldLocation reorder_location_ = QUDA_CPU_FIELD_LOCATION;
#endif

int zeroCopy = 0;

namespace quda {

  bool cudaColorSpinorField::initGhostFaceBuffer = false;
  size_t cudaColorSpinorField::ghostFaceBytes = 0;

  cudaColorSpinorField::cudaColorSpinorField(const ColorSpinorParam &param) : 
    ColorSpinorField(param), alloc(false), init(true), texInit(false),
    ghostTexInit(false), ghost_field_tex{nullptr,nullptr}, bufferMessageHandler(0)
  {
    // this must come before create
    if (param.create == QUDA_REFERENCE_FIELD_CREATE) {
      v = param.v;
      norm = param.norm;
    }

    create(param.create);

    if  (param.create == QUDA_NULL_FIELD_CREATE) {
      // do nothing
    } else if (param.create == QUDA_ZERO_FIELD_CREATE) {
      zero();
    } else if (param.create == QUDA_REFERENCE_FIELD_CREATE) {
      // do nothing
    } else if (param.create == QUDA_COPY_FIELD_CREATE) {
      errorQuda("not implemented");
    }
  }

  cudaColorSpinorField::cudaColorSpinorField(const cudaColorSpinorField &src) : 
    ColorSpinorField(src), alloc(false), init(true), texInit(false),
    ghostTexInit(false), ghost_field_tex{nullptr,nullptr}, bufferMessageHandler(0)
  {
    create(QUDA_COPY_FIELD_CREATE);
    copySpinorField(src);
  }

  // creates a copy of src, any differences defined in param
  cudaColorSpinorField::cudaColorSpinorField(const ColorSpinorField &src, 
					     const ColorSpinorParam &param) :
    ColorSpinorField(src), alloc(false), init(true), texInit(false),
    ghostTexInit(false), ghost_field_tex{nullptr,nullptr}, bufferMessageHandler(0)
  {
    // can only overide if we are not using a reference or parity special case
    if (param.create != QUDA_REFERENCE_FIELD_CREATE || 
	(param.create == QUDA_REFERENCE_FIELD_CREATE && 
	 src.SiteSubset() == QUDA_FULL_SITE_SUBSET && 
	 param.siteSubset == QUDA_PARITY_SITE_SUBSET && 
	 typeid(src) == typeid(cudaColorSpinorField) ) || 
         (param.create == QUDA_REFERENCE_FIELD_CREATE && (param.is_composite || param.is_component))) {
      reset(param);
    } else {
      errorQuda("Undefined behaviour"); // else silent bug possible?
    }

    // This must be set before create is called
    if (param.create == QUDA_REFERENCE_FIELD_CREATE) {
      if (typeid(src) == typeid(cudaColorSpinorField)) {
	v = (void*)src.V();
	norm = (void*)src.Norm();
      } else {
	errorQuda("Cannot reference a non-cuda field");
      }

      if (composite_descr.is_component && !(src.SiteSubset() == QUDA_FULL_SITE_SUBSET && this->SiteSubset() == QUDA_PARITY_SITE_SUBSET)) 
      {//setup eigenvector form the set
        v    = (void*)((char*)v    + composite_descr.id*bytes);         
        norm = (void*)((char*)norm + composite_descr.id*norm_bytes);         
      }
    }

    create(param.create);

    if (param.create == QUDA_NULL_FIELD_CREATE) {
      // do nothing
    } else if (param.create == QUDA_ZERO_FIELD_CREATE) {
      zero();
    } else if (param.create == QUDA_COPY_FIELD_CREATE) {
      copySpinorField(src);
    } else if (param.create == QUDA_REFERENCE_FIELD_CREATE) {
      // do nothing
    } else {
      errorQuda("CreateType %d not implemented", param.create);
    }

  }

  cudaColorSpinorField::cudaColorSpinorField(const ColorSpinorField &src) 
    : ColorSpinorField(src), alloc(false), init(true), texInit(false),
      ghostTexInit(false), ghost_field_tex{nullptr,nullptr}, bufferMessageHandler(0)
  {
    create(QUDA_COPY_FIELD_CREATE);
    copySpinorField(src);
  }

  ColorSpinorField& cudaColorSpinorField::operator=(const ColorSpinorField &src) {
    if (typeid(src) == typeid(cudaColorSpinorField)) {
      *this = (dynamic_cast<const cudaColorSpinorField&>(src));
    } else if (typeid(src) == typeid(cpuColorSpinorField)) {
      *this = (dynamic_cast<const cpuColorSpinorField&>(src));
    } else {
      errorQuda("Unknown input ColorSpinorField %s", typeid(src).name());
    }
    return *this;
  }

  cudaColorSpinorField& cudaColorSpinorField::operator=(const cudaColorSpinorField &src) {
    if (&src != this) {
      // keep current attributes unless unset
      if (!ColorSpinorField::init) { // note this will turn a reference field into a regular field
	destroy();
	destroyComms(); // not sure if this necessary
	ColorSpinorField::operator=(src);
	create(QUDA_COPY_FIELD_CREATE);
      }
      copySpinorField(src);
    }
    return *this;
  }

  cudaColorSpinorField& cudaColorSpinorField::operator=(const cpuColorSpinorField &src) {
    // keep current attributes unless unset
    if (!ColorSpinorField::init) { // note this will turn a reference field into a regular field
      destroy();
      ColorSpinorField::operator=(src);
      create(QUDA_COPY_FIELD_CREATE);
    }
    loadSpinorField(src);
    return *this;
  }

  cudaColorSpinorField::~cudaColorSpinorField() {
    destroyComms();
    destroy();
  }

  void cudaColorSpinorField::create(const QudaFieldCreate create) {

    if (siteSubset == QUDA_FULL_SITE_SUBSET && siteOrder != QUDA_EVEN_ODD_SITE_ORDER) {
      errorQuda("Subset not implemented");
    }

    if (create != QUDA_REFERENCE_FIELD_CREATE) {
      v = pool_device_malloc(bytes);
      if (precision == QUDA_HALF_PRECISION) norm = pool_device_malloc(norm_bytes);
      alloc = true;
    }

    if (siteSubset == QUDA_FULL_SITE_SUBSET) {
      if(composite_descr.is_composite && (create != QUDA_REFERENCE_FIELD_CREATE)) {
	if(composite_descr.dim <= 0) errorQuda("\nComposite size is not defined\n");
	  
        ColorSpinorParam param;
        param.siteSubset = QUDA_FULL_SITE_SUBSET;
        param.nDim = nDim;
        memcpy(param.x, x, nDim*sizeof(int));
        param.create = QUDA_REFERENCE_FIELD_CREATE;
        param.v = v;
        param.norm = norm;
        param.is_composite   = false;
        param.composite_dim  = 0;
        param.is_component = true;

        components.reserve(composite_descr.dim);
        for(int cid = 0; cid < composite_descr.dim; cid++) {
	  param.component_id = cid;
	  components.push_back(new cudaColorSpinorField(*this, param));
        }
      } else {
        // create the associated even and odd subsets
        ColorSpinorParam param;
        param.siteSubset = QUDA_PARITY_SITE_SUBSET;
        param.nDim = nDim;
        memcpy(param.x, x, nDim*sizeof(int));
        param.x[0] /= 2; // set single parity dimensions
        param.create = QUDA_REFERENCE_FIELD_CREATE;
        param.v = v;
        param.norm = norm;
        param.is_composite  = false;
        param.composite_dim = 0;
        param.is_component  = composite_descr.is_component;
        param.component_id  = composite_descr.id;
        even = new cudaColorSpinorField(*this, param);
        odd = new cudaColorSpinorField(*this, param);

        // need this hackery for the moment (need to locate the odd pointers half way into the full field)
        (dynamic_cast<cudaColorSpinorField*>(odd))->v = (void*)((char*)v + bytes/2);
        if (precision == QUDA_HALF_PRECISION) 
	  (dynamic_cast<cudaColorSpinorField*>(odd))->norm = (void*)((char*)norm + norm_bytes/2);

#ifdef USE_TEXTURE_OBJECTS
        dynamic_cast<cudaColorSpinorField*>(even)->destroyTexObject();
        dynamic_cast<cudaColorSpinorField*>(even)->createTexObject();
        dynamic_cast<cudaColorSpinorField*>(odd)->destroyTexObject();
        dynamic_cast<cudaColorSpinorField*>(odd)->createTexObject();
#endif
      }
    } else { //siteSubset == QUDA_PARITY_SITE_SUBSET

      //! setup an object for selected eigenvector (the 1st one as a default):
      if (composite_descr.is_composite && (create != QUDA_REFERENCE_FIELD_CREATE)) 
      {
         if(composite_descr.dim <= 0) errorQuda("\nComposite size is not defined\n");
         //if(bytes > 1811939328) warningQuda("\nCUDA API probably won't be able to create texture object for the eigenvector set... Object size is : %u bytes\n", bytes);
         if (getVerbosity() == QUDA_DEBUG_VERBOSE) printfQuda("\nEigenvector set constructor...\n");
         // create the associated even and odd subsets
         ColorSpinorParam param;
         param.siteSubset = QUDA_PARITY_SITE_SUBSET;
         param.nDim = nDim;
         memcpy(param.x, x, nDim*sizeof(int));
         param.create = QUDA_REFERENCE_FIELD_CREATE;
         param.v = v;
         param.norm = norm;
         param.is_composite   = false;
         param.composite_dim  = 0;
         param.is_component = true;
         //reserve eigvector set
         components.reserve(composite_descr.dim);
         //setup volume, [real_]length and stride for a single eigenvector
         for(int cid = 0; cid < composite_descr.dim; cid++)
         {
            param.component_id = cid;
            components.push_back(new cudaColorSpinorField(*this, param));

#ifdef USE_TEXTURE_OBJECTS //(a lot of texture objects...)
            dynamic_cast<cudaColorSpinorField*>(components[cid])->destroyTexObject();
            dynamic_cast<cudaColorSpinorField*>(components[cid])->createTexObject();
#endif
         }
      }
    }

    if (create != QUDA_REFERENCE_FIELD_CREATE) {
      if (siteSubset != QUDA_FULL_SITE_SUBSET) {
	zeroPad();
      } else if(!composite_descr.is_composite) {
	(dynamic_cast<cudaColorSpinorField*>(even))->zeroPad();
	(dynamic_cast<cudaColorSpinorField*>(odd))->zeroPad();
      } else { //temporary hack for the full spinor field sets, manual zeroPad for each component:
	for(int cid = 0; cid < composite_descr.dim; cid++) {
	  (dynamic_cast<cudaColorSpinorField&>(components[cid]->Even())).zeroPad();
	  (dynamic_cast<cudaColorSpinorField&>(components[cid]->Odd())).zeroPad();
	}
      }
    }

#ifdef USE_TEXTURE_OBJECTS
    if (!composite_descr.is_composite || composite_descr.is_component)
      createTexObject();
#endif
  }

#ifdef USE_TEXTURE_OBJECTS
  void cudaColorSpinorField::createTexObject() {

    if (isNative()) {
      if (texInit) errorQuda("Already bound textures");
      
      // create the texture for the field components
      
      hipChannelFormatDesc desc;
      memset(&desc, 0, sizeof(hipChannelFormatDesc));
      if (precision == QUDA_SINGLE_PRECISION) desc.f = hipChannelFormatKindFloat;
      else desc.f = hipChannelFormatKindSigned; // half is short, double is int2
      
      // staggered and coarse fields in half and single are always two component
      if ( (nSpin == 1 || nSpin == 2) && (precision == QUDA_HALF_PRECISION || precision == QUDA_SINGLE_PRECISION)) {
	desc.x = 8*precision;
	desc.y = 8*precision;
	desc.z = 0;
	desc.w = 0;
      } else { // all others are four component (double2 is spread across int4)
	desc.x = (precision == QUDA_DOUBLE_PRECISION) ? 32 : 8*precision;
	desc.y = (precision == QUDA_DOUBLE_PRECISION) ? 32 : 8*precision;
	desc.z = (precision == QUDA_DOUBLE_PRECISION) ? 32 : 8*precision;
	desc.w = (precision == QUDA_DOUBLE_PRECISION) ? 32 : 8*precision;
      }
      
      hipResourceDesc resDesc;
      memset(&resDesc, 0, sizeof(resDesc));
      resDesc.resType = hipResourceTypeLinear;
      resDesc.res.linear.devPtr = v;
      resDesc.res.linear.desc = desc;
      resDesc.res.linear.sizeInBytes = bytes;
      
      hipTextureDesc texDesc;
      memset(&texDesc, 0, sizeof(texDesc));
      if (precision == QUDA_HALF_PRECISION) texDesc.readMode = hipReadModeNormalizedFloat;
      else texDesc.readMode = hipReadModeElementType;
      
      hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL);

      // create the texture for the norm components
      if (precision == QUDA_HALF_PRECISION) {
	hipChannelFormatDesc desc;
	memset(&desc, 0, sizeof(hipChannelFormatDesc));
	desc.f = hipChannelFormatKindFloat;
	desc.x = 8*QUDA_SINGLE_PRECISION; desc.y = 0; desc.z = 0; desc.w = 0;
	
	hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeLinear;
	resDesc.res.linear.devPtr = norm;
	resDesc.res.linear.desc = desc;
	resDesc.res.linear.sizeInBytes = norm_bytes;
	
	hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.readMode = hipReadModeElementType;
	
	hipCreateTextureObject(&texNorm, &resDesc, &texDesc, NULL);
      }
      
      texInit = true;

      checkCudaError();
    }
  }

  void cudaColorSpinorField::createGhostTexObject() {
    // create the ghost texture object
    if (isNative() && ghost_bytes) {
      if (ghostTexInit) errorQuda("Already bound ghost texture");

      for (int b=0; b<2; b++) {
	hipChannelFormatDesc desc;
	memset(&desc, 0, sizeof(hipChannelFormatDesc));
	if (precision == QUDA_SINGLE_PRECISION) desc.f = hipChannelFormatKindFloat;
	else desc.f = hipChannelFormatKindSigned; // half is short, double is int2

	// staggered and coarse fields in half and single are always two component
	if ( (nSpin == 1 || nSpin == 2) && (precision == QUDA_HALF_PRECISION || precision == QUDA_SINGLE_PRECISION)) {
	  desc.x = 8*precision;
	  desc.y = 8*precision;
	  desc.z = 0;
	  desc.w = 0;
	} else { // all others are four component (double2 is spread across int4)
	  desc.x = (precision == QUDA_DOUBLE_PRECISION) ? 32 : 8*precision;
	  desc.y = (precision == QUDA_DOUBLE_PRECISION) ? 32 : 8*precision;
	  desc.z = (precision == QUDA_DOUBLE_PRECISION) ? 32 : 8*precision;
	  desc.w = (precision == QUDA_DOUBLE_PRECISION) ? 32 : 8*precision;
	}

	hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeLinear;
	resDesc.res.linear.devPtr = ghost_field[b];
	resDesc.res.linear.desc = desc;
	resDesc.res.linear.sizeInBytes = ghost_bytes;

	hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	if (precision == QUDA_HALF_PRECISION) texDesc.readMode = hipReadModeNormalizedFloat;
	else texDesc.readMode = hipReadModeElementType;

	hipCreateTextureObject(&ghostTex[b], &resDesc, &texDesc, NULL);

	if (precision == QUDA_HALF_PRECISION) {
	  hipChannelFormatDesc desc;
	  memset(&desc, 0, sizeof(hipChannelFormatDesc));
	  desc.f = hipChannelFormatKindFloat;
	  desc.x = 8*QUDA_SINGLE_PRECISION; desc.y = 0; desc.z = 0; desc.w = 0;

	  hipResourceDesc resDesc;
	  memset(&resDesc, 0, sizeof(resDesc));
	  resDesc.resType = hipResourceTypeLinear;
	  resDesc.res.linear.devPtr = ghost_field[b];
	  resDesc.res.linear.desc = desc;
	  resDesc.res.linear.sizeInBytes = ghost_bytes;

	  hipTextureDesc texDesc;
	  memset(&texDesc, 0, sizeof(texDesc));
	  texDesc.readMode = hipReadModeElementType;

	  hipCreateTextureObject(&ghostTexNorm[b], &resDesc, &texDesc, NULL);
	}

	ghost_field_tex[b] = ghost_field[b];
      } // buffer index

      ghostTexInit = true;

      checkCudaError();
    }
  }

  void cudaColorSpinorField::destroyTexObject() {
    if (isNative() && texInit) {
      hipDestroyTextureObject(tex);
      if (ghost_bytes) {
	hipDestroyTextureObject(ghostTex[0]);
	hipDestroyTextureObject(ghostTex[1]);
      }
      if (precision == QUDA_HALF_PRECISION) {
        hipDestroyTextureObject(texNorm);
        if (ghost_bytes) {
	  hipDestroyTextureObject(ghostTexNorm[0]);
	  hipDestroyTextureObject(ghostTexNorm[1]);
	}
      }
      texInit = false;
    }
  }

  void cudaColorSpinorField::destroyGhostTexObject() {
    if (isNative() && ghostTexInit) {
      hipDestroyTextureObject(ghostTex[0]);
      hipDestroyTextureObject(ghostTex[1]);
      if (precision == QUDA_HALF_PRECISION) {
	hipDestroyTextureObject(ghostTexNorm[0]);
	hipDestroyTextureObject(ghostTexNorm[1]);
      }
      ghostTexInit = false;
    }
  }
#endif

  void cudaColorSpinorField::destroy() {
    if (alloc) {
      pool_device_free(v);
      if (precision == QUDA_HALF_PRECISION) pool_device_free(norm);
      alloc = false;
    }

    if (composite_descr.is_composite) 
    {
       CompositeColorSpinorField::iterator vec;
       for(vec = components.begin(); vec != components.end(); vec++) delete *vec;
    } 

    if ((siteSubset == QUDA_FULL_SITE_SUBSET && !composite_descr.is_composite) || (siteSubset == QUDA_FULL_SITE_SUBSET && composite_descr.is_component)) {
      delete even;
      delete odd;
    }

#ifdef USE_TEXTURE_OBJECTS
    if (!composite_descr.is_composite || composite_descr.is_component)
      destroyTexObject();
#endif

  }

  // cuda's floating point format, IEEE-754, represents the floating point
  // zero as 4 zero bytes
  void cudaColorSpinorField::zero() {
    hipMemsetAsync(v, 0, bytes, streams[Nstream-1]);
    if (precision == QUDA_HALF_PRECISION) hipMemsetAsync(norm, 0, norm_bytes, streams[Nstream-1]);
  }

  void cudaColorSpinorField::zeroPad() {
    size_t pad_bytes = (stride - volume) * precision * fieldOrder;
    int Npad = nColor * nSpin * 2 / fieldOrder;

    if (composite_descr.is_composite && !composite_descr.is_component){//we consider the whole eigenvector set:
      Npad      *= composite_descr.dim;
      pad_bytes /= composite_descr.dim;
    }

    size_t pitch = ((!composite_descr.is_composite || composite_descr.is_component) ? stride : composite_descr.stride)*fieldOrder*precision;
    char   *dst  = (char*)v + ((!composite_descr.is_composite || composite_descr.is_component) ? volume : composite_descr.volume)*fieldOrder*precision;
    if (pad_bytes) hipMemset2D(dst, pitch, 0, pad_bytes, Npad);

    //for (int i=0; i<Npad; i++) {
    //  if (pad_bytes) hipMemset((char*)v + (volume + i*stride)*fieldOrder*precision, 0, pad_bytes);
    //}
  }

  void cudaColorSpinorField::copy(const cudaColorSpinorField &src) {
    checkField(*this, src);
    if (this->GammaBasis() != src.GammaBasis()) errorQuda("cannot call this copy with different basis");
    blas::copy(*this, src);
  }

  void cudaColorSpinorField::copySpinorField(const ColorSpinorField &src) {
    
    // src is on the device and is native
    if (typeid(src) == typeid(cudaColorSpinorField) && 
	isNative() && dynamic_cast<const cudaColorSpinorField &>(src).isNative() &&
	this->GammaBasis() == src.GammaBasis()) {
      copy(dynamic_cast<const cudaColorSpinorField&>(src));
    } else if (typeid(src) == typeid(cudaColorSpinorField)) {
      copyGenericColorSpinor(*this, src, QUDA_CUDA_FIELD_LOCATION);
    } else if (typeid(src) == typeid(cpuColorSpinorField)) { // src is on the host
      loadSpinorField(src);
    } else {
      errorQuda("Unknown input ColorSpinorField %s", typeid(src).name());
    }
  } 

  void cudaColorSpinorField::loadSpinorField(const ColorSpinorField &src) {

    if (reorder_location_ == QUDA_CPU_FIELD_LOCATION &&typeid(src) == typeid(cpuColorSpinorField)) {
      void *buffer = pool_pinned_malloc(bytes + norm_bytes);
      memset(buffer, 0, bytes+norm_bytes); // FIXME (temporary?) bug fix for padding

      copyGenericColorSpinor(*this, src, QUDA_CPU_FIELD_LOCATION, buffer, 0, static_cast<char*>(buffer)+bytes, 0);

      qudaMemcpy(v, buffer, bytes, hipMemcpyHostToDevice);
      qudaMemcpy(norm, static_cast<char*>(buffer)+bytes, norm_bytes, hipMemcpyHostToDevice);

      pool_pinned_free(buffer);
    } else if (typeid(src) == typeid(cudaColorSpinorField)) {
      copyGenericColorSpinor(*this, src, QUDA_CUDA_FIELD_LOCATION);
    } else {
      void *Src=nullptr, *srcNorm=nullptr, *buffer=nullptr;
      if (!zeroCopy) {
	resizeBufferDevice(src.Bytes()+src.NormBytes());
	Src = bufferDevice;
	srcNorm = (char*)bufferDevice + src.Bytes();
	qudaMemcpy(Src, src.V(), src.Bytes(), hipMemcpyHostToDevice);
	qudaMemcpy(srcNorm, src.Norm(), src.NormBytes(), hipMemcpyHostToDevice);
      } else {
	buffer = pool_pinned_malloc(src.Bytes()+src.NormBytes());
	memcpy(buffer, src.V(), src.Bytes());
	memcpy(static_cast<char*>(buffer)+src.Bytes(), src.Norm(), src.NormBytes());

	hipHostGetDevicePointer(&Src, buffer, 0);
	srcNorm = (void*)((char*)Src + src.Bytes());
      }

      hipMemset(v, 0, bytes); // FIXME (temporary?) bug fix for padding
      copyGenericColorSpinor(*this, src, QUDA_CUDA_FIELD_LOCATION, 0, Src, 0, srcNorm);

      if (zeroCopy) pool_pinned_free(buffer);
    }

    return;
  }


  void cudaColorSpinorField::saveSpinorField(ColorSpinorField &dest) const {

    if (reorder_location_ == QUDA_CPU_FIELD_LOCATION &&	typeid(dest) == typeid(cpuColorSpinorField)) {
      void *buffer = pool_pinned_malloc(bytes+norm_bytes);
      qudaMemcpy(buffer, v, bytes, hipMemcpyDeviceToHost);
      qudaMemcpy(static_cast<char*>(buffer)+bytes, norm, norm_bytes, hipMemcpyDeviceToHost);

      copyGenericColorSpinor(dest, *this, QUDA_CPU_FIELD_LOCATION, 0, buffer, 0, static_cast<char*>(buffer)+bytes);
      pool_pinned_free(buffer);
    } else if (typeid(dest) == typeid(cudaColorSpinorField)) {
      copyGenericColorSpinor(dest, *this, QUDA_CUDA_FIELD_LOCATION);
    } else {
      void *dst=nullptr, *dstNorm=nullptr, *buffer=nullptr;
      if (!zeroCopy) {
	resizeBufferDevice(dest.Bytes()+dest.NormBytes());
	dst = bufferDevice;
	dstNorm = (char*)bufferDevice+dest.Bytes();
      } else {
	buffer = pool_pinned_malloc(dest.Bytes()+dest.NormBytes());
	hipHostGetDevicePointer(&dst, buffer, 0);
	dstNorm = (char*)dst+dest.Bytes();
      }
      copyGenericColorSpinor(dest, *this, QUDA_CUDA_FIELD_LOCATION, dst, v, dstNorm, 0);

      if (!zeroCopy) {
	qudaMemcpy(dest.V(), dst, dest.Bytes(), hipMemcpyDeviceToHost);
	qudaMemcpy(dest.Norm(), dstNorm, dest.NormBytes(), hipMemcpyDeviceToHost);
      } else {
	memcpy(dest.V(), buffer, dest.Bytes());
	memcpy(dest.Norm(), static_cast<char*>(buffer) + dest.Bytes(), dest.NormBytes());
      }

      if (zeroCopy) pool_pinned_free(buffer);
    }

    return;
  }

  void cudaColorSpinorField::allocateGhostBuffer(int nFace) {

    if (!comm_partitioned()) {
      for (int i=0; i<4; i++) ghost_face_bytes[i] = 0;
      return;
    }

    createGhostZone(nFace);

    // temporary work around until the ghost buffer for fine and
    // coarse grid are merged: this ensures we reset the fine ghost
    // buffer if the coarse grid operator allocates a ghost buffer
    // that is larger than the fine grid operator
    static size_t ghostFaceBytes_ = 0;

    // only allocate if not already allocated or buffer required is bigger than previously
    if ( !initGhostFaceBuffer || ghost_bytes > ghostFaceBytes || ghost_bytes > ghostFaceBytes_) {

      if (initGhostFaceBuffer) {
#ifdef USE_TEXTURE_OBJECTS
	destroyGhostTexObject();
#endif
	if (ghost_bytes) {
	  for (int b=0; b<2; b++) {
	    device_pinned_free(ghost_field[b]);
	    device_free(ghostFaceBuffer[b]);
	    host_free(ghost_pinned_h[b]);
	  }
	}
      }

      if (ghost_bytes > 0) {
	for (int b=0; b<2; ++b) {
	  // gpu receive buffer (use pinned allocator to avoid this being redirected, e.g., by QDPJIT)
	  ghost_field[b] = device_pinned_malloc(ghost_bytes);

	  // gpu send buffset
	  ghostFaceBuffer[b] = device_malloc(ghost_bytes);

	  // pinned buffer used for sending and receiving
	  ghost_pinned_h[b] = pinned_malloc(2*ghost_bytes);

	  // set the matching device-mapper pointer
	  hipHostGetDevicePointer(&ghost_pinned_d[b], ghost_pinned_h[b], 0);
	}

	initGhostFaceBuffer = true;
	ghostFaceBytes = ghost_bytes;
	ghostFaceBytes_ = ghost_bytes;
      }

      LatticeField::ghost_field_reset = true; // this signals that we must reset the IPC comms
    }

#ifdef USE_TEXTURE_OBJECTS
    // ghost texture is per object
    if (ghost_field_tex[0] != ghost_field[0] || ghost_field_tex[1] != ghost_field[1]) destroyGhostTexObject();
    if (!ghostTexInit) createGhostTexObject();
#endif
  }

  void cudaColorSpinorField::allocateGhostBuffer(void *send_buf[], void *recv_buf[]) const
  {
    int num_faces = 1;
    if (nSpin == 1) num_faces = 3; // staggered

    int spinor_size = 2*nSpin*nColor*precision;

    // resize face only if requested size is larger than previously allocated one
    size_t faceBytes = 0;
    for (int i=0; i<nDimComms; i++) {
      if (comm_dim_partitioned(i)) faceBytes += 2*siteSubset*num_faces*surfaceCB[i]*spinor_size;
    }

    static size_t ghostFaceBytes_ = 0;

    if (!initGhostFaceBuffer || faceBytes > ghostFaceBytes || faceBytes > ghostFaceBytes_) {

      if (initGhostFaceBuffer) {
	for (int b=0; b<2; ++b) device_free(ghostFaceBuffer[b]);
      }

      if (faceBytes > 0) {
	for (int b=0; b<2; ++b) ghostFaceBuffer[b] = device_malloc(faceBytes);
	initGhostFaceBuffer = true;
	ghostFaceBytes = faceBytes;
	ghostFaceBytes_ = faceBytes;
      }

    }

    size_t offset = 0;
    for (int i=0; i<nDimComms; i++) {
      if (comm_dim_partitioned(i)) {
	// use first buffer for recv and second for send
	recv_buf[2*i+0] = static_cast<void*>((static_cast<char*>(ghostFaceBuffer[0]) + offset));
	send_buf[2*i+0] = static_cast<void*>((static_cast<char*>(ghostFaceBuffer[1]) + offset));
	offset += siteSubset*num_faces*surfaceCB[i]*spinor_size;

	recv_buf[2*i+1] = static_cast<void*>((static_cast<char*>(ghostFaceBuffer[0]) + offset));
	send_buf[2*i+1] = static_cast<void*>((static_cast<char*>(ghostFaceBuffer[1]) + offset));
	offset += siteSubset*num_faces*surfaceCB[i]*spinor_size;
      } else {
	recv_buf[2*i+0] = nullptr;
	recv_buf[2*i+1] = nullptr;
	send_buf[2*i+0] = nullptr;
	send_buf[2*i+1] = nullptr;
      }
    }

  }

  void cudaColorSpinorField::freeGhostBuffer(void)
  {
    destroyIPCComms();

    if (!initGhostFaceBuffer) return;
  
    for (int b=0; b<2; b++) {
      // free receive buffer
      if (ghost_field[b]) device_pinned_free(ghost_field[b]);
      ghost_field[b] = nullptr;

      // free send buffer
      if (ghostFaceBuffer[b]) device_free(ghostFaceBuffer[b]);
      ghostFaceBuffer[b] = nullptr;

      // free pinned memory buffers
      if (ghost_pinned_h[b]) host_free(ghost_pinned_h[b]);
      ghost_pinned_h[b] = nullptr;
      ghost_pinned_d[b] = nullptr;
    }
    initGhostFaceBuffer = false;
  }

  // pack the ghost zone into a contiguous buffer for communications
  void cudaColorSpinorField::packGhost(const int nFace, const QudaParity parity, 
                                       const int dim, const QudaDirection dir,
				       const int dagger, hipStream_t *stream, 
				       void *buffer, bool zero_copy, double a, double b)
  {
#ifdef MULTI_GPU
    int face_num;
    if (dir == QUDA_BACKWARDS) {
      face_num = 0;
    }else if (dir == QUDA_FORWARDS) {
      face_num = 1;
    }else{
      face_num = 2;
    }
    void *packBuffer = buffer ? buffer : ghostFaceBuffer[bufferIndex];
    packFace(packBuffer, *this, zero_copy, nFace, dagger, parity, dim, face_num, *stream, a, b);
#else
    errorQuda("packGhost not built on single-GPU build");
#endif
  }
 
  // send the ghost zone to the host
  void cudaColorSpinorField::sendGhost(void *ghost_spinor, const int nFace, const int dim, 
				       const QudaDirection dir, const int dagger, 
				       hipStream_t *stream) {

#ifdef MULTI_GPU
    int Nvec = (nSpin == 1 || precision == QUDA_DOUBLE_PRECISION) ? 2 : 4;
    int Nint = (nColor * nSpin * 2) / (nSpin == 4 ? 2 : 1);  // (spin proj.) degrees of freedom
    int Npad = Nint / Nvec; // number Nvec buffers we have
    
    if (dim !=3 || getKernelPackT() || getTwistPack()) { // use kernels to pack into contiguous buffers then a single hipMemcpy

      size_t bytes = nFace*Nint*ghostFace[dim]*precision;

      if (precision == QUDA_HALF_PRECISION) bytes += nFace*ghostFace[dim]*sizeof(float);

      void* gpu_buf = 
	(dir == QUDA_BACKWARDS) ? this->backGhostFaceBuffer[bufferIndex][dim] : this->fwdGhostFaceBuffer[bufferIndex][dim];

      hipMemcpyAsync(ghost_spinor, gpu_buf, bytes, hipMemcpyDeviceToHost, *stream);

    } else if (this->TwistFlavor() != QUDA_TWIST_NONDEG_DOUBLET) { // do multiple cudaMemcpys

      const int x4 = nDim==5 ? x[4] : 1;
      const int Nt_minus1_offset = (volumeCB - nFace*ghostFace[3])/x4; // N_t -1 = Vh-Vsh

      int offset = 0;
      if (nSpin == 1) {
	offset = (dir == QUDA_BACKWARDS) ? 0 : Nt_minus1_offset;
      } else if (nSpin == 4) {
	// !dagger: send lower components backwards, send upper components forwards
	// dagger: send upper components backwards, send lower components forwards
	bool upper = dagger ? true : false; // Fwd is !Back  
	if (dir == QUDA_FORWARDS) upper = !upper;
	int lower_spin_offset = Npad*stride;
	if (upper) offset = (dir == QUDA_BACKWARDS ? 0 : Nt_minus1_offset);
	else offset = lower_spin_offset + (dir == QUDA_BACKWARDS ? 0 : Nt_minus1_offset);
      }
    
      size_t len = nFace*(ghostFace[3]/x4)*Nvec*precision;
      size_t dpitch = x4*len;
      size_t spitch = stride*Nvec*precision;

      // QUDA Memcpy NPad's worth. 
      //  -- Dest will point to the right beginning PAD. 
      //  -- Each Pad has size Nvec*Vsh Floats. 
      //  --  There is Nvec*Stride Floats from the start of one PAD to the start of the next
      for (int s=0; s<x4; s++) { // loop over multiple 4-d volumes (if they exist)
	void *dst = (char*)ghost_spinor + s*len;
	void *src = (char*)v + (offset + s*(volumeCB/x4))*Nvec*precision;
	hipMemcpy2DAsync(dst, dpitch, src, spitch, len, Npad, hipMemcpyDeviceToHost, *stream);

	if (precision == QUDA_HALF_PRECISION) {
	  size_t len = nFace*(ghostFace[3]/x4)*sizeof(float);
	  int norm_offset = (dir == QUDA_BACKWARDS) ? 0 : Nt_minus1_offset*sizeof(float);
	  void *dst = (char*)ghost_spinor + nFace*Nint*ghostFace[3]*precision + s*len;
	  void *src = (char*)norm + norm_offset + s*(volumeCB/x4)*sizeof(float);
	  hipMemcpyAsync(dst, src, len, hipMemcpyDeviceToHost, *stream);
	}
      }
    }else{
      int flavorVolume = volume / 2;
      int flavorTFace  = ghostFace[3] / 2;
      int flavor1_Nt_minus1_offset = (flavorVolume - flavorTFace);
      int flavor2_Nt_minus1_offset = (volume - flavorTFace);
      int flavor1_offset = 0;
      int flavor2_offset = 0;
      // !dagger: send lower components backwards, send upper components forwards
      // dagger: send upper components backwards, send lower components forwards
      bool upper = dagger ? true : false; // Fwd is !Back
      if (dir == QUDA_FORWARDS) upper = !upper;
      int lower_spin_offset = Npad*stride;//ndeg tm: stride=2*flavor_volume+pad
      if (upper) {
        flavor1_offset = (dir == QUDA_BACKWARDS ? 0 : flavor1_Nt_minus1_offset);
        flavor2_offset = (dir == QUDA_BACKWARDS ? flavorVolume : flavor2_Nt_minus1_offset);
      }else{
        flavor1_offset = lower_spin_offset + (dir == QUDA_BACKWARDS ? 0 : flavor1_Nt_minus1_offset);
        flavor2_offset = lower_spin_offset + (dir == QUDA_BACKWARDS ? flavorVolume : flavor2_Nt_minus1_offset);
      }

      // QUDA Memcpy NPad's worth.
      //  -- Dest will point to the right beginning PAD.
      //  -- Each Pad has size Nvec*Vsh Floats.
      //  --  There is Nvec*Stride Floats from the start of one PAD to the start of the next

      void *dst = (char*)ghost_spinor;
      void *src = (char*)v + flavor1_offset*Nvec*precision;
      size_t len = flavorTFace*Nvec*precision;
      size_t spitch = stride*Nvec*precision;//ndeg tm: stride=2*flavor_volume+pad
      size_t dpitch = 2*len;
      hipMemcpy2DAsync(dst, dpitch, src, spitch, len, Npad, hipMemcpyDeviceToHost, *stream);
      dst = (char*)ghost_spinor+len;
      src = (char*)v + flavor2_offset*Nvec*precision;
      hipMemcpy2DAsync(dst, dpitch, src, spitch, len, Npad, hipMemcpyDeviceToHost, *stream);

      if (precision == QUDA_HALF_PRECISION) {
        int Nt_minus1_offset = (flavorVolume - flavorTFace);
        int norm_offset = (dir == QUDA_BACKWARDS) ? 0 : Nt_minus1_offset*sizeof(float);
	void *dst = (char*)ghost_spinor + Nint*ghostFace[3]*precision;
	void *src = (char*)norm + norm_offset;
        size_t dpitch = flavorTFace*sizeof(float);
        size_t spitch = flavorVolume*sizeof(float);
	hipMemcpy2DAsync(dst, dpitch, src, spitch, flavorTFace*sizeof(float), 2, hipMemcpyDeviceToHost, *stream);
      }
    }
#else
    errorQuda("sendGhost not built on single-GPU build");
#endif

  }



  void cudaColorSpinorField::unpackGhost(const void* ghost_spinor, const int nFace, 
					 const int dim, const QudaDirection dir, 
					 const int dagger, hipStream_t* stream) 
  {
    int Nint = (nColor * nSpin * 2) / (nSpin == 4 ? 2 : 1);  // (spin proj.) degrees of freedom

    int len = nFace*ghostFace[dim]*Nint*precision;
    const void *src = ghost_spinor;
  
    int ghost_offset = (dir == QUDA_BACKWARDS) ? ghostOffset[dim][0] : ghostOffset[dim][1];
    void *ghost_dst = (char*)ghost_field[bufferIndex] + precision*ghost_offset;

    if (precision == QUDA_HALF_PRECISION) len += nFace*ghostFace[dim]*sizeof(float);

    hipMemcpyAsync(ghost_dst, src, len, hipMemcpyHostToDevice, *stream);
  }




   // pack the ghost zone into a contiguous buffer for communications
  void cudaColorSpinorField::packGhostExtended(const int nFace, const int R[], const QudaParity parity,
					       const int dim, const QudaDirection dir,
					       const int dagger, hipStream_t *stream,
					       void *buffer, bool zero_copy)
  {
#ifdef MULTI_GPU
    int face_num;
    if (dir == QUDA_BACKWARDS) {
      face_num = 0;
    }else if (dir == QUDA_FORWARDS) {
      face_num = 1;
    }else{
      face_num = 2;
    }
    void *packBuffer = buffer ? buffer : ghostFaceBuffer[bufferIndex];
    packFaceExtended(packBuffer, *this, zero_copy, nFace, R, dagger, parity, dim, face_num, *stream);
#else
    errorQuda("packGhostExtended not built on single-GPU build");
#endif

  }


  // copy data from host buffer into boundary region of device field
  void cudaColorSpinorField::unpackGhostExtended(const void* ghost_spinor, const int nFace, const QudaParity parity,
                                                 const int dim, const QudaDirection dir, 
                                                 const int dagger, hipStream_t* stream, bool zero_copy)
  {

     
     
    // First call the regular unpackGhost routine to copy data into the `usual' ghost-zone region 
    // of the data array 
    unpackGhost(ghost_spinor, nFace, dim, dir, dagger, stream);

    // Next step is to copy data from the ghost zone back to the interior region
    int Nint = (nColor * nSpin * 2) / (nSpin == 4 ? 2 : 1); // (spin proj.) degrees of freedom

    int len = nFace*ghostFace[dim]*Nint;
    int offset = length + ghostOffset[dim][0];
    offset += (dir == QUDA_BACKWARDS) ? 0 : len;

#ifdef MULTI_GPU
    const int face_num = 2;
    const bool unpack = true;
    const int R[4] = {0,0,0,0};
    packFaceExtended(ghostFaceBuffer[bufferIndex], *this, zero_copy, nFace, R, dagger, parity, dim, face_num, *stream, unpack);
#else
    errorQuda("unpackGhostExtended not built on single-GPU build");
#endif
  }


  hipStream_t *stream;

  void cudaColorSpinorField::createComms(int nFace) {

    allocateGhostBuffer(nFace); // allocate the ghost buffer if not yet allocated

    // ascertain if this instance needs its comms buffers to be updated
    bool comms_reset = ghost_field_reset || // FIXME add send buffer check
      (my_face[0] != ghost_pinned_h[0]) || (my_face[1] != ghost_pinned_h[1]) || // pinned buffers
      (ghost_field_tex[0] != ghost_field[0]) || (ghost_field_tex[1] != ghost_field[1]); // receive buffers

    if (!initComms || comms_reset) {

      // if we are requesting a new number of faces destroy and start over
      destroyComms();

      if (siteSubset != QUDA_PARITY_SITE_SUBSET) errorQuda("Only supports single parity fields");

      int Nint = nColor * nSpin * 2 / (nSpin == 4 ? 2 : 1); // number of internal degrees of freedom

      for (int i=0; i<nDimComms; i++) { // compute size of ghost buffers required
	if (!commDimPartitioned(i)) { ghost_face_bytes[i] = 0; continue; }
	ghost_face_bytes[i] = nFace*ghostFace[i]*Nint*precision;
	if (precision == QUDA_HALF_PRECISION) ghost_face_bytes[i] += nFace*ghostFace[i]*sizeof(float);
      }

      // initialize the ghost pinned buffers
      for (int b=0; b<2; b++) {
	my_face[b] = ghost_pinned_h[b];
	my_face_d[b] = ghost_pinned_d[b];
	from_face[b] = static_cast<char*>(my_face[b]) + ghost_bytes;
	from_face_d[b] = static_cast<char*>(my_face_d[b]) + ghost_bytes;
      }

      // initialize the ghost receive pointers
      for (int i=0; i<nDimComms; ++i) {
	if (commDimPartitioned(i)) {
	  for (int b=0; b<2; b++) {
	    ghost[b][i] = static_cast<char*>(ghost_field[b]) + ghostOffset[i][0]*precision;
	    if (precision == QUDA_HALF_PRECISION)
	      ghostNorm[b][i] = static_cast<char*>(ghost_field[b]) + ghostNormOffset[i][0]*QUDA_SINGLE_PRECISION;
	  }
	}
      }

      // initialize ghost send pointers
      size_t offset = 0;
      for (int i=0; i<nDimComms; i++) {
	if (!commDimPartitioned(i)) continue;

	for (int b=0; b<2; ++b) {
	  backGhostFaceBuffer[b][i] = (void*)(((char*)ghostFaceBuffer[b]) + offset);

	  my_back_face[b][i] = static_cast<char*>(my_face[b]) + offset;
	  from_back_face[b][i] = static_cast<char*>(from_face[b]) + offset;

	  my_back_face_rdma[b][i] = backGhostFaceBuffer[b][i];
	  from_back_face_rdma[b][i] = static_cast<char*>(ghost_field[b]) + ghostOffset[i][0]*precision;
	} // loop over b
	offset += ghost_face_bytes[i];

	for (int b=0; b<2; ++b) {
	  fwdGhostFaceBuffer[b][i] = (void*)(((char*)ghostFaceBuffer[b]) + offset);

	  my_fwd_face[b][i] = static_cast<char*>(my_face[b]) + offset;
	  from_fwd_face[b][i] = static_cast<char*>(from_face[b]) + offset;

	  my_fwd_face_rdma[b][i] = fwdGhostFaceBuffer[b][i];
	  from_fwd_face_rdma[b][i] = static_cast<char*>(ghost_field[b]) + ghostOffset[i][1]*precision;
	} // loop over b
	offset += ghost_face_bytes[i];

      } // loop over dimension

      // initialize the message handlers
      for (int i=0; i<nDimComms; i++) {
	if (!commDimPartitioned(i)) continue;

	for (int b=0; b<2; ++b) {
	  mh_send_fwd[b][i] = comm_declare_send_relative(my_fwd_face[b][i], i, +1, ghost_face_bytes[i]);
	  mh_send_back[b][i] = comm_declare_send_relative(my_back_face[b][i], i, -1, ghost_face_bytes[i]);

	  mh_recv_fwd[b][i] = comm_declare_receive_relative(from_fwd_face[b][i], i, +1, ghost_face_bytes[i]);
	  mh_recv_back[b][i] = comm_declare_receive_relative(from_back_face[b][i], i, -1, ghost_face_bytes[i]);

	  mh_send_rdma_fwd[b][i] = comm_declare_send_relative(my_fwd_face_rdma[b][i], i, +1, ghost_face_bytes[i]);
	  mh_send_rdma_back[b][i] = comm_declare_send_relative(my_back_face_rdma[b][i], i, -1, ghost_face_bytes[i]);

	  mh_recv_rdma_fwd[b][i] = comm_declare_receive_relative(from_fwd_face_rdma[b][i], i, +1, ghost_face_bytes[i]);
	  mh_recv_rdma_back[b][i] = comm_declare_receive_relative(from_back_face_rdma[b][i], i, -1, ghost_face_bytes[i]);
	} // loop over b

      } // loop over dimension
     
      initComms = true;
      checkCudaError();
    }

    if (LatticeField::ghost_field_reset) destroyIPCComms();
    createIPCComms();
  }

  void cudaColorSpinorField::destroyComms()
  {
    if (initComms) {

      for (int b=0; b<2; ++b) {
	for (int i=0; i<nDimComms; i++) {
	  if (commDimPartitioned(i)) {
	    if (mh_recv_fwd[b][i]) comm_free(mh_recv_fwd[b][i]);
	    if (mh_recv_back[b][i]) comm_free(mh_recv_back[b][i]);
	    if (mh_send_fwd[b][i]) comm_free(mh_send_fwd[b][i]);
	    if (mh_send_back[b][i]) comm_free(mh_send_back[b][i]);

	    if (mh_recv_rdma_fwd[b][i]) comm_free(mh_recv_rdma_fwd[b][i]);
	    if (mh_recv_rdma_back[b][i]) comm_free(mh_recv_rdma_back[b][i]);
	    if (mh_send_rdma_fwd[b][i]) comm_free(mh_send_rdma_fwd[b][i]);
	    if (mh_send_rdma_back[b][i]) comm_free(mh_send_rdma_back[b][i]);
	  }
	}
      } // loop over b

      initComms = false;
      checkCudaError();
    }

  }

  void cudaColorSpinorField::streamInit(hipStream_t *stream_p) {
    stream = stream_p;
  }


  void cudaColorSpinorField::pack(int nFace, int parity, int dagger, hipStream_t *stream_p, 
				  bool zero_copy, double a, double b) {

    createComms(nFace); // must call this first

    stream = stream_p;
    
    const int dim=-1; // pack all partitioned dimensions
 
    if (zero_copy) {
      packGhost(nFace, (QudaParity)parity, dim, QUDA_BOTH_DIRS, dagger, &stream[0], my_face_d[bufferIndex], true, a, b);
    } else {
      packGhost(nFace, (QudaParity)parity, dim, QUDA_BOTH_DIRS, dagger, &stream[Nstream-1], 0, false, a, b);
    }
  }

  void cudaColorSpinorField::pack(int nFace, int parity, int dagger, int stream_idx, 
				  bool zeroCopyPack, double a, double b) {

    createComms(nFace); // must call this first

    const int dim=-1; // pack all partitioned dimensions
 
    if (zeroCopyPack) {
      packGhost(nFace, (QudaParity)parity, dim, QUDA_BOTH_DIRS, dagger, &stream[stream_idx], my_face_d[bufferIndex], true, a, b);
    } else {
      packGhost(nFace, (QudaParity)parity, dim, QUDA_BOTH_DIRS, dagger, &stream[stream_idx], 0, false, a, b);
    }
  }

  void cudaColorSpinorField::packExtended(const int nFace, const int R[], const int parity, 
                                          const int dagger, const int dim,
                                          hipStream_t *stream_p, const bool zero_copy) {

    createComms(nFace); // must call this first

    stream = stream_p;
 
    if (zero_copy) {
      packGhostExtended(nFace, R, (QudaParity)parity, dim, QUDA_BOTH_DIRS, dagger, &stream[0], my_face_d[bufferIndex], true);
    }else{
      packGhostExtended(nFace, R, (QudaParity)parity, dim, QUDA_BOTH_DIRS, dagger, &stream[Nstream-1], 0, false);
    }
  }
                                                      


  void cudaColorSpinorField::gather(int nFace, int dagger, int dir, hipStream_t* stream_p)
  {
    int dim = dir/2;

    // If stream_p != 0, use pack_stream, else use the stream array
    hipStream_t *pack_stream = (stream_p) ? stream_p : stream+dir;

    if (dir%2 == 0) {
      // backwards copy to host
      if (comm_peer2peer_enabled(0,dim)) return;

      sendGhost(my_back_face[bufferIndex][dim], nFace, dim, QUDA_BACKWARDS, dagger, pack_stream);
    } else {
      // forwards copy to host
      if (comm_peer2peer_enabled(1,dim)) return;

      sendGhost(my_fwd_face[bufferIndex][dim], nFace, dim, QUDA_FORWARDS, dagger, pack_stream);
    }
  }


  void cudaColorSpinorField::recvStart(int nFace, int dir, int dagger, hipStream_t* stream_p, bool gdr) {

    int dim = dir/2;
    if (!commDimPartitioned(dim)) return;

    //fprintf(stderr, "%s dir=%d stream_p=%p\n", __func__, dir, stream_p);
    hipStream_t stream = stream_p ? *stream_p : NULL;

    if (dir%2 == 0) { // sending backwards
      if (comm_peer2peer_enabled(1,dim)) {
        // receive from the processor in the +1 direction
        comm_start(mh_recv_p2p_fwd[bufferIndex][dim]);
      } else if (gdr) {
        // Prepost receive
        comm_start_on_stream(mh_recv_rdma_fwd[bufferIndex][dim], stream);
      } else {
        // Prepost receive
        comm_start_on_stream(mh_recv_fwd[bufferIndex][dim], stream);
      }
    } else { //sending forwards
      // Prepost receive
      if (comm_peer2peer_enabled(0,dim)) {
        comm_start(mh_recv_p2p_back[bufferIndex][dim]);
      } else if (gdr) {
        comm_start_on_stream(mh_recv_rdma_back[bufferIndex][dim], stream);
      } else {
        comm_start_on_stream(mh_recv_back[bufferIndex][dim], stream);
      }
    }
  }


  void cudaColorSpinorField::sendStart(int nFace, int d, int dagger, hipStream_t* stream_p, bool gdr) {

    int dim = d/2;
    int dir = d%2;
    if (!commDimPartitioned(dim)) return;

    //fprintf(stderr, "%s dir=%d stream_p=%p\n", __func__, dir, stream_p);

    int Nvec = (nSpin == 1 || precision == QUDA_DOUBLE_PRECISION) ? 2 : 4;
    int Nint = (nColor * nSpin * 2)/(nSpin == 4 ? 2 : 1); // (spin proj.) degrees of freedom
    int Npad = Nint/Nvec;

    if (!comm_peer2peer_enabled(dir,dim)) {
      hipStream_t stream = stream_p ? *stream_p : NULL;
      if (dir == 0)
        if (gdr) comm_start_on_stream(mh_send_rdma_back[bufferIndex][dim], stream);
        else comm_start_on_stream(mh_send_back[bufferIndex][dim], stream);
      else
        if (gdr) comm_start_on_stream(mh_send_rdma_fwd[bufferIndex][dim], stream);
        else comm_start_on_stream(mh_send_fwd[bufferIndex][dim], stream);
    } else { // doing peer-to-peer
      hipStream_t *copy_stream = (stream_p) ? stream_p : stream + d;

      // all goes here
      void* ghost_dst = (dir == 0) ? (void*)((char*)(backGhostSendDest[bufferIndex][dim]) + precision*ghostOffset[dim][1]) :
	(void*)((char*)(fwdGhostSendDest[bufferIndex][dim]) + precision*ghostOffset[dim][0]);
      void *ghost_norm_dst = (dir == 0) ? static_cast<char*>(backGhostSendDest[bufferIndex][dim]) + QUDA_SINGLE_PRECISION*ghostNormOffset[dim][1] :
	static_cast<char*>(fwdGhostSendDest[bufferIndex][dim]) + QUDA_SINGLE_PRECISION*ghostNormOffset[dim][0];

      if (dim != 3 || getKernelPackT()) {

	hipMemcpyAsync(ghost_dst,
			dir == 0 ? backGhostFaceBuffer[bufferIndex][dim] : fwdGhostFaceBuffer[bufferIndex][dim],
			ghost_face_bytes[dim],
			hipMemcpyDeviceToDevice,
			*copy_stream); // copy to forward processor

      } else if (this->TwistFlavor() != QUDA_TWIST_NONDEG_DOUBLET) {

	const int x4 = nDim==5 ? x[4] : 1;
	const int Nt_minus_offset = (volumeCB - nFace*ghostFace[3])/x4;

	int offset = 0;
	if (nSpin == 1) {
	  offset = (dir == 0) ? 0 : Nt_minus_offset;
	} else if (nSpin == 4) {
	  // !dagger: send lower components backwards, send upper components forwards
	  // dagger: send upper components backwards, send lower components forwards
	  bool upper = dagger ? true : false;
	  if (dir == 1) upper = !upper;
	  int lower_spin_offset = Npad*stride;
	  if (dir == 0) {
	    offset = upper ? 0 : lower_spin_offset;
	  } else {
	    offset = (upper) ? Nt_minus_offset : lower_spin_offset + Nt_minus_offset;
	  }
	}

	size_t len = nFace*(ghostFace[3]/x4)*Nvec*precision;
	size_t dpitch = x4*len;
	size_t spitch = stride*Nvec*precision;

	for (int s=0; s<x4; s++) {
	  void *dst = (char*)ghost_dst + s*len;
	  void *src = (char*)v + (offset + s*(volumeCB/x4))*Nvec*precision;
	  // start the copy
	  hipMemcpy2DAsync(dst, dpitch, src, spitch, len, Npad, hipMemcpyDeviceToDevice, *copy_stream);

	  if (precision == QUDA_HALF_PRECISION) {
	    size_t len = nFace*(ghostFace[3]/x4)*sizeof(float);
	    int norm_offset = (dir == 0) ? 0 : Nt_minus_offset*sizeof(float);
	    void *dst = (char*)ghost_norm_dst + s*len;
	    void *src = static_cast<char*>(norm) + norm_offset + s*(volumeCB/x4)*sizeof(float);
	    hipMemcpyAsync(dst, src, len, hipMemcpyDeviceToDevice, *copy_stream);
	  }
	}
      } else { // twisted doublet
	int flavorVolume = volume / 2;
	int flavorTFace  = ghostFace[3] / 2;
	int flavor1_Nt_minus1_offset = (flavorVolume - flavorTFace);
	int flavor2_Nt_minus1_offset = (volume - flavorTFace);
	int flavor1_offset = 0;
	int flavor2_offset = 0;
	// !dagger: send lower components backwards, send upper components forwards
	// dagger: send upper components backwards, send lower components forwards
	bool upper = dagger ? true : false; // Fwd is !Back
	if (dir == 1) upper = !upper;
	int lower_spin_offset = Npad*stride;//ndeg tm: stride=2*flavor_volume+pad
	if (upper) {
	  flavor1_offset = (dir == 0 ? 0 : flavor1_Nt_minus1_offset);
	  flavor2_offset = (dir == 0 ? flavorVolume : flavor2_Nt_minus1_offset);
	}else{
	  flavor1_offset = lower_spin_offset + (dir == 0 ? 0 : flavor1_Nt_minus1_offset);
	  flavor2_offset = lower_spin_offset + (dir == 0 ? flavorVolume : flavor2_Nt_minus1_offset);
	}

	// QUDA Memcpy NPad's worth.
	//  -- Dest will point to the right beginning PAD.
	//  -- Each Pad has size Nvec*Vsh Floats.
	//  --  There is Nvec*Stride Floats from the start of one PAD to the start of the next

	void *src = static_cast<char*>(v) + flavor1_offset*Nvec*precision;
	size_t len = flavorTFace*Nvec*precision;
	size_t spitch = stride*Nvec*precision;//ndeg tm: stride=2*flavor_volume+pad
	size_t dpitch = 2*len;
	hipMemcpy2DAsync(ghost_dst, dpitch, src, spitch, len, Npad, hipMemcpyDeviceToDevice, *copy_stream);

	src = static_cast<char*>(v) + flavor2_offset*Nvec*precision;
	hipMemcpy2DAsync(static_cast<char*>(ghost_dst)+len, dpitch, src, spitch, len, Npad, hipMemcpyDeviceToDevice, *copy_stream);

	if (precision == QUDA_HALF_PRECISION) {
	  int norm_offset = (dir == 0) ? 0 : flavor1_Nt_minus1_offset*sizeof(float);
	  void *src = static_cast<char*>(norm) + norm_offset;
	  size_t dpitch = flavorTFace*sizeof(float);
	  size_t spitch = flavorVolume*sizeof(float);
	  hipMemcpy2DAsync(ghost_norm_dst, dpitch, src, spitch, flavorTFace*sizeof(float), 2, hipMemcpyDeviceToDevice, *copy_stream);
	}
      }

      if (dir == 0) {
	// record the event
	hipEventRecord(ipcCopyEvent[bufferIndex][0][dim], *copy_stream);
	// send to the propcessor in the -1 direction
	comm_start(mh_send_p2p_back[bufferIndex][dim]);
      } else {
	hipEventRecord(ipcCopyEvent[bufferIndex][1][dim], *copy_stream);
	// send to the processor in the +1 direction
	comm_start(mh_send_p2p_fwd[bufferIndex][dim]);
      }
    }
  }

  void cudaColorSpinorField::commsStart(int nFace, int dir, int dagger, hipStream_t* stream_p, bool gdr) {
    recvStart(nFace, dir, dagger, stream_p, gdr);
    sendStart(nFace, dir, dagger, stream_p, gdr);
  }


  static bool complete_recv_fwd[QUDA_MAX_DIM] = { };
  static bool complete_recv_back[QUDA_MAX_DIM] = { };
  static bool complete_send_fwd[QUDA_MAX_DIM] = { };
  static bool complete_send_back[QUDA_MAX_DIM] = { };

  int cudaColorSpinorField::commsQuery(int nFace, int dir, int dagger, hipStream_t *stream_p, bool gdr) {

    int dim = dir/2;
    if (!commDimPartitioned(dim)) return 0;

    if (dir%2==0) {

      if (comm_peer2peer_enabled(1,dim)) {
	if (!complete_recv_fwd[dim]) complete_recv_fwd[dim] = comm_query(mh_recv_p2p_fwd[bufferIndex][dim]);
      } else if (gdr) {
	if (!complete_recv_fwd[dim]) complete_recv_fwd[dim] = comm_query(mh_recv_rdma_fwd[bufferIndex][dim]);
      } else {
	if (!complete_recv_fwd[dim]) complete_recv_fwd[dim] = comm_query(mh_recv_fwd[bufferIndex][dim]);
      }

      if (comm_peer2peer_enabled(0,dim)) {
	if (!complete_send_back[dim]) complete_send_back[dim] = comm_query(mh_send_p2p_back[bufferIndex][dim]);
      } else if (gdr) {
	if (!complete_send_back[dim]) complete_send_back[dim] = comm_query(mh_send_rdma_back[bufferIndex][dim]);
      } else {
	if (!complete_send_back[dim]) complete_send_back[dim] = comm_query(mh_send_back[bufferIndex][dim]);
      }

      if (complete_recv_fwd[dim] && complete_send_back[dim]) {
	complete_recv_fwd[dim] = false;
	complete_send_back[dim] = false;
	return 1;
      }

    } else { // dir%2 == 1

      if (comm_peer2peer_enabled(0,dim)) {
	if (!complete_recv_back[dim]) complete_recv_back[dim] = comm_query(mh_recv_p2p_back[bufferIndex][dim]);
      } else if (gdr) {
	if (!complete_recv_back[dim]) complete_recv_back[dim] = comm_query(mh_recv_rdma_back[bufferIndex][dim]);
      } else {
	if (!complete_recv_back[dim]) complete_recv_back[dim] = comm_query(mh_recv_back[bufferIndex][dim]);
      }

      if (comm_peer2peer_enabled(1,dim)) {
	if (!complete_send_fwd[dim]) complete_send_fwd[dim] = comm_query(mh_send_p2p_fwd[bufferIndex][dim]);
      } else if (gdr) {
	if (!complete_send_fwd[dim]) complete_send_fwd[dim] = comm_query(mh_send_rdma_fwd[bufferIndex][dim]);
      } else {
	if (!complete_send_fwd[dim]) complete_send_fwd[dim] = comm_query(mh_send_fwd[bufferIndex][dim]);
      }

      if (complete_recv_back[dim] && complete_send_fwd[dim]) {
	complete_recv_back[dim] = false;
	complete_send_fwd[dim] = false;
	return 1;
      }

    }

    return 0;
  }

void cudaColorSpinorField::commsWait(int nFace, int dir, int dagger, hipStream_t *stream_p, bool gdr, unsigned mask) {
    int dim = dir / 2;
    if (!commDimPartitioned(dim)) return;

    //fprintf(stderr, "%s dir=%d stream_p=%p\n", __func__, dir, stream_p);
    hipStream_t stream = stream_p ? *stream_p : NULL;

    if (dir%2==0) {
      if (mask & wait_recv) {
      if (comm_peer2peer_enabled(1,dim)) {
	comm_wait(mh_recv_p2p_fwd[bufferIndex][dim]);
	hipEventSynchronize(ipcRemoteCopyEvent[bufferIndex][1][dim]);
      } else if (gdr) {
        comm_wait_on_stream(mh_recv_rdma_fwd[bufferIndex][dim], stream);
      } else {
        comm_wait_on_stream(mh_recv_fwd[bufferIndex][dim], stream);
      }
      }
      if (mask & wait_send) {
      if (comm_peer2peer_enabled(0,dim)) {
	comm_wait(mh_send_p2p_back[bufferIndex][dim]);
	hipEventSynchronize(ipcCopyEvent[bufferIndex][0][dim]);
      } else if (gdr) {
        comm_wait_on_stream(mh_send_rdma_back[bufferIndex][dim], stream);
      } else {
        comm_wait_on_stream(mh_send_back[bufferIndex][dim], stream);
      }
      }
    } else {
      if (mask & wait_recv) {
      if (comm_peer2peer_enabled(0,dim)) {
	comm_wait(mh_recv_p2p_back[bufferIndex][dim]);
	hipEventSynchronize(ipcRemoteCopyEvent[bufferIndex][0][dim]);
      } else if (gdr) {
	comm_wait(mh_recv_rdma_back[bufferIndex][dim]);
      } else {
        comm_wait_on_stream(mh_recv_back[bufferIndex][dim], stream);
      }
      }
      if (mask & wait_send) {
      if (comm_peer2peer_enabled(1,dim)) {
	comm_wait(mh_send_p2p_fwd[bufferIndex][dim]);
	hipEventSynchronize(ipcCopyEvent[bufferIndex][1][dim]);
      } else if (gdr) {
	comm_wait(mh_send_rdma_fwd[bufferIndex][dim]);
      } else {
        comm_wait_on_stream(mh_send_fwd[bufferIndex][dim], stream);
      }
      }
    }

    return;
  }

  void cudaColorSpinorField::scatter(int nFace, int dagger, int dir, hipStream_t* stream_p)
  {
    int dim = dir/2;
    if (!commDimPartitioned(dim)) return;

    if (dir%2==0) {// receive from forwards
      if (comm_peer2peer_enabled(1,dim)) return;

      unpackGhost(from_fwd_face[bufferIndex][dim], nFace, dim, QUDA_FORWARDS, dagger, stream_p);
    } else { // receive from backwards
      if (comm_peer2peer_enabled(0,dim)) return;

      unpackGhost(from_back_face[bufferIndex][dim], nFace, dim, QUDA_BACKWARDS, dagger, stream_p);
    }
  }



  void cudaColorSpinorField::scatter(int nFace, int dagger, int dir)
  {
    int dim = dir/2;
    if (!commDimPartitioned(dim)) return;

    if (dir%2==0) {// receive from forwards
      if (comm_peer2peer_enabled(1,dim)) return;

      unpackGhost(from_fwd_face[bufferIndex][dim], nFace, dim, QUDA_FORWARDS, dagger, &stream[2*dim+0]);
    } else { // receive from backwards
      if (comm_peer2peer_enabled(0,dim)) return;

      unpackGhost(from_back_face[bufferIndex][dim], nFace, dim, QUDA_BACKWARDS, dagger, &stream[2*dim+1]);
    }
  }

  
  void cudaColorSpinorField::scatterExtended(int nFace, int parity, int dagger, int dir)
  {
    bool zero_copy = false;
    int dim = dir/2;
    if (!commDimPartitioned(dim)) return;
    if (dir%2==0) {// receive from forwards
      unpackGhostExtended(from_fwd_face[bufferIndex][dim], nFace, static_cast<QudaParity>(parity), dim, QUDA_FORWARDS, dagger, &stream[2*dim/*+0*/], zero_copy);
    } else { // receive from backwards
      unpackGhostExtended(from_back_face[bufferIndex][dim], nFace, static_cast<QudaParity>(parity),  dim, QUDA_BACKWARDS, dagger, &stream[2*dim/*+1*/], zero_copy);
    }
  }
 
  void cudaColorSpinorField::exchangeGhost(QudaParity parity, int dagger) const {
    void **send = static_cast<void**>(safe_malloc(nDimComms * 2 * sizeof(void*)));

    // allocate ghost buffer if not yet allocated
    allocateGhostBuffer(send, ghost_fixme);

    genericPackGhost(send, *this, parity, dagger);

    int nFace = (nSpin == 1) ? 3 : 1;
    exchange(ghost_fixme, send, nFace);

    host_free(send);
  }

  std::ostream& operator<<(std::ostream &out, const cudaColorSpinorField &a) {
    out << (const ColorSpinorField&)a;
    out << "v = " << a.v << std::endl;
    out << "norm = " << a.norm << std::endl;
    out << "alloc = " << a.alloc << std::endl;
    out << "init = " << a.init << std::endl;
    return out;
  }

//! for composite fields:
  cudaColorSpinorField& cudaColorSpinorField::Component(const int idx) const {
    
    if (this->IsComposite()) {
      if (idx < this->CompositeDim()) {//setup eigenvector form the set
        return *(dynamic_cast<cudaColorSpinorField*>(components[idx])); 
      }
      else{
        errorQuda("Incorrect component index...");
      }
    }
    errorQuda("Cannot get requested component");
    exit(-1);
  }

//copyCuda currently cannot not work with set of spinor fields..
  void cudaColorSpinorField::CopySubset(cudaColorSpinorField &dst, const int range, const int first_element) const{
#if 0
    if (first_element < 0) errorQuda("\nError: trying to set negative first element.\n");
    if (siteSubset == QUDA_PARITY_SITE_SUBSET && this->EigvId() == -1) {
      if (first_element == 0 && range == this->EigvDim())
      {
        if (range != dst.EigvDim())errorQuda("\nError: eigenvector range to big.\n");
        checkField(dst, *this);
        copyCuda(dst, *this);
      }
      else if ((first_element+range) < this->EigvDim()) 
      {//setup eigenvector subset

        cudaColorSpinorField *eigv_subset;

        ColorSpinorParam param;

        param.nColor = nColor;
        param.nSpin = nSpin;
        param.twistFlavor = twistFlavor;
        param.precision = precision;
        param.nDim = nDim;
        param.pad = pad;
        param.siteSubset = siteSubset;
        param.siteOrder = siteOrder;
        param.fieldOrder = fieldOrder;
        param.gammaBasis = gammaBasis;
        memcpy(param.x, x, nDim*sizeof(int));
        param.create = QUDA_REFERENCE_FIELD_CREATE;
 
        param.eigv_dim  = range;
        param.eigv_id   = -1;
        param.v = (void*)((char*)v + first_element*eigv_bytes);
        param.norm = (void*)((char*)norm + first_element*eigv_norm_bytes);

        eigv_subset = new cudaColorSpinorField(param);

        //Not really needed:
        eigv_subset->eigenvectors.reserve(param.eigv_dim);
        for (int id = first_element; id < (first_element+range); id++)
        {
            param.eigv_id = id;
            eigv_subset->eigenvectors.push_back(new cudaColorSpinorField(*this, param));
        }
        checkField(dst, *eigv_subset);
        copyCuda(dst, *eigv_subset);

        delete eigv_subset;
      } else {
        errorQuda("Incorrect eigenvector dimension...");
      }
    } else{
      errorQuda("Eigenvector must be a parity spinor");
      exit(-1);
    }
#endif
  }

  void cudaColorSpinorField::getTexObjectInfo() const
  {
#ifdef USE_TEXTURE_OBJECTS
    printfQuda("\nPrint texture info for the field:\n");
    std::cout << *this;
    hipResourceDesc resDesc;
    //memset(&resDesc, 0, sizeof(resDesc));
    hipGetTextureObjectResourceDesc(&resDesc, this->Tex());
    printfQuda("\nDevice pointer: %p\n", resDesc.res.linear.devPtr);
    printfQuda("\nVolume (in bytes): %lu\n", resDesc.res.linear.sizeInBytes);
    if (resDesc.resType == hipResourceTypeLinear) printfQuda("\nResource type: linear \n");
#endif
  }

  void cudaColorSpinorField::Source(const QudaSourceType sourceType, const int st, const int s, const int c) {
    ColorSpinorParam param(*this);
    param.fieldOrder = QUDA_SPACE_SPIN_COLOR_FIELD_ORDER;
    param.location = QUDA_CPU_FIELD_LOCATION;
    param.create = QUDA_NULL_FIELD_CREATE;

    cpuColorSpinorField tmp(param);
    tmp.Source(sourceType, st, s, c);
    *this = tmp;
  }

  void cudaColorSpinorField::PrintVector(unsigned int i) {
    ColorSpinorParam param(*this);
    param.fieldOrder = QUDA_SPACE_SPIN_COLOR_FIELD_ORDER;
    param.location = QUDA_CPU_FIELD_LOCATION;
    param.create = QUDA_NULL_FIELD_CREATE;

    cpuColorSpinorField tmp(param);
    tmp = *this;
    tmp.PrintVector(i);
  }


} // namespace quda
